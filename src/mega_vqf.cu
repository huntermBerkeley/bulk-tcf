#include "hip/hip_runtime.h"


#ifndef mega_vqf_C
#define mega_vqf_C



#include "include/mega_vqf.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "include/megablock.cuh"

#include <iostream>

#include <fstream>
#include <assert.h>


#define MAX_FILL 28

__device__ void mega_vqf::lock_block(int warpID, uint64_t lock){


	// if (warpID == 0){

	// 	while(atomicCAS(locks + lock, 0,1) != 0);	
	// }
	// __syncwarp();

	//TODO: turn me back on
	blocks[lock].lock(warpID);
}

__device__ void mega_vqf::unlock_block(int warpID, uint64_t lock){


	// if (warpID == 0){

	// 	while(atomicCAS(locks + lock, 1,0) != 1);	

	// }

	// __syncwarp();

	blocks[lock].unlock(warpID);
}

__device__ void mega_vqf::lock_blocks(int warpID, uint64_t lock1, uint64_t lock2){


	if (lock1 < lock2){

		lock_block(warpID, lock1);
		lock_block(warpID, lock2);
		//while(atomicCAS(locks + lock2, 0,1) == 1);

	} else {


		lock_block(warpID, lock2);
		lock_block(warpID, lock1);
		
	}

	


}

__device__ void mega_vqf::unlock_blocks(int warpID, uint64_t lock1, uint64_t lock2){


	if (lock1 > lock2){

		unlock_block(warpID, lock1);
		unlock_block(warpID, lock2);
		
	} else {

		unlock_block(warpID, lock2);
		unlock_block(warpID, lock1);
	}
	

}

__device__ bool mega_vqf::insert(int warpID, uint64_t hash){

   uint64_t block_index = (hash >> TAG_BITS) % num_blocks;



   //this will generate a mask and get the tag bits
   uint64_t tag = hash & ((1ULL << TAG_BITS) -1);
   uint64_t alt_block_index = (((hash ^ (tag * 0x5bd1e995)) % (num_blocks*SLOTS_PER_BLOCK)) >> TAG_BITS) % num_blocks;

   // assert(block_index < num_blocks);


   //external locks
   //blocks[block_index].extra_lock(block_index);
 	
 	lock_block(warpID, block_index);
 	int side_fill = blocks[block_index].get_fill();

 	//side_fill < 20 ||

 	if (block_index == alt_block_index){


 		if (side_fill < MAX_FILL){

 			blocks[block_index].insert(warpID, hash);


 			unlock_block(warpID, block_index);

 			return true;

 		} else {

 			unlock_block(warpID, block_index);
 			return false;
 		}

 		


 		

 	}


 	unlock_block(warpID, block_index);


 	lock_blocks(warpID, block_index, alt_block_index);

   int fill_main = blocks[block_index].get_fill();

   int fill_alt = blocks[alt_block_index].get_fill();


   bool toReturn = false;

   if (fill_main < fill_alt){


   	unlock_block(warpID, alt_block_index);



   	//if (fill_main < SLOTS_PER_BLOCK-1){
   	if (fill_main < MAX_FILL){
   		blocks[block_index].insert(warpID, hash);

   		toReturn = true;

   	}

   	unlock_block(warpID, block_index);


   } else {

   	unlock_block(warpID, block_index);

   	if (fill_alt < MAX_FILL){

	   	blocks[alt_block_index].insert(warpID, hash);

	   	toReturn = true;

	   }

	   unlock_block(warpID, alt_block_index);

   }



 	//unlock_blocks(block_index, alt_block_index);


   return toReturn;





}


__device__ bool mega_vqf::query(int warpID, uint64_t hash){

	uint64_t block_index = (hash >> TAG_BITS) % num_blocks;

   //this will generate a mask and get the tag bits
   uint64_t tag = hash & ((1ULL << TAG_BITS) -1);
   uint64_t alt_block_index = (((hash ^ (tag * 0x5bd1e995)) % (num_blocks*SLOTS_PER_BLOCK)) >> TAG_BITS) % num_blocks;

   if (block_index == alt_block_index){

   	lock_block(warpID, block_index);



   	bool found = blocks[block_index].query(warpID, hash);

   	unlock_block(warpID, block_index);

   	return found;


   }

   lock_blocks(warpID, block_index, alt_block_index);


   bool found = blocks[block_index].query(warpID, hash) || blocks[alt_block_index].query(warpID, hash);


   unlock_blocks(warpID, block_index, alt_block_index);

   return found;

}


__device__ bool mega_vqf::remove(int warpID, uint64_t hash){


	uint64_t block_index = (hash >> TAG_BITS) % num_blocks;

   //this will generate a mask and get the tag bits
   uint64_t tag = hash & ((1ULL << TAG_BITS) -1);
   uint64_t alt_block_index = (((hash ^ (tag * 0x5bd1e995)) % (num_blocks*SLOTS_PER_BLOCK)) >> TAG_BITS) % num_blocks;


   lock_block(warpID, block_index);

   bool found = blocks[block_index].remove(warpID, hash);

   unlock_block(warpID, block_index);

   //copy could be deleted from this instance

   if (found){
   	return true;
   }

   lock_block(warpID, alt_block_index);

   found = blocks[alt_block_index].remove(warpID, hash);


   unlock_block(warpID, alt_block_index);

   return found;

}


// __device__ bool mega_vqf::insert(uint64_t hash){

//    uint64_t block_index = (hash >> TAG_BITS) % num_blocks;



//    //this will generate a mask and get the tag bits
//    uint64_t tag = hash & ((1ULL << TAG_BITS) -1);
//    uint64_t alt_block_index = (((hash ^ (tag * 0x5bd1e995)) % (num_blocks*SLOTS_PER_BLOCK)) >> TAG_BITS) % num_blocks;

//    assert(block_index < num_blocks);


//    //external locks
//    //blocks[block_index].extra_lock(block_index);
   
//    while(atomicCAS(locks + block_index, 0, 1) == 1);



//    int fill_main = blocks[block_index].get_fill();


//    if (fill_main >= SLOTS_PER_BLOCK-1){

//    	while(atomicCAS(locks + block_index, 0, 1) == 0);
//    	//blocks[block_index].unlock();

//    	return false;
//    }

//    if (fill_main < .75 * SLOTS_PER_BLOCK || block_index == alt_block_index){
//    	blocks[block_index].insert(tag);

   	

//    	int new_fill = blocks[block_index].get_fill();
//    	if (new_fill != fill_main+1){
//    		printf("Broken Fill: Block %llu, old %d new %d\n", block_index, fill_main, new_fill);
//    		assert(blocks[block_index].get_fill() == fill_main+1);
//    	}


//    	while(atomicCAS(locks + block_index, 1, 0) == 0);
//    	//blocks[block_index].unlock();
//    	return true;
//    }


//    while(atomicCAS(locks + block_index, 1, 0) == 0);

//    lock_blocks(block_index, alt_block_index);


//    //need to grab other block

//    //blocks[alt_block_index].extra_lock(alt_block_index);
//    while(atomicCAS(locks + alt_block_index, 0, 1) == 1);

//    int fill_alt = blocks[alt_block_index].get_fill();

//    //any larger and we can't protect metadata
//    if (fill_alt >=  SLOTS_PER_BLOCK-1){
// //   	blocks[block_index.unlock()]

//    	unlock_blocks(block_index, alt_block_index);
//    	//blocks[alt_block_index].unlock();
//    	//blocks[block_index].unlock();
//    	return false;
//    }


//    //unlock main
//    if (fill_main > fill_alt ){

//    	while(atomicCAS(locks + block_index, 1, 0) == 0);
//    	//blocks[block_index].unlock();

//    	blocks[alt_block_index].insert(tag);
//    	assert(blocks[alt_block_index].get_fill() == fill_alt+1);

//    	int new_fill = blocks[alt_block_index].get_fill();
//    	if (new_fill != fill_alt+1){
//    		printf("Broken Fill: Block %llu, old %d new %d\n", alt_block_index, fill_alt, new_fill);
//    		assert(blocks[alt_block_index].get_fill() == fill_alt+1);
//    	}

//    	while(atomicCAS(locks + alt_block_index, 1, 0) == 0);
//    	//blocks[alt_block_index].unlock();


//    } else {

//    	while(atomicCAS(locks + alt_block_index, 1, 0) == 0);
//    	//blocks[alt_block_index].unlock();
//    	blocks[block_index].insert(tag);

//    	int new_fill = blocks[block_index].get_fill();
//    	if (new_fill != fill_main+1){
//    		printf("Broken Fill: Block %llu, old %d new %d\n", block_index, fill_main, new_fill);
//    		assert(blocks[block_index].get_fill() == fill_main+1);
//    	}

//    	while(atomicCAS(locks + block_index, 1, 0) == 0);
//    	//blocks[block_index].unlock();

//    }


  
//    return true;



//}


__global__ void mega_vqf_block_setup(mega_vqf * mega_vqf){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid >= mega_vqf->num_blocks) return;

	mega_vqf->blocks[tid].setup();

}

__host__ mega_vqf * build_mega_vqf(uint64_t nitems){


	//this seems weird but whatever
	uint64_t num_blocks = (nitems -1)/SLOTS_PER_BLOCK + 1;


	printf("Bytes used: %llu for %llu blocks.\n", num_blocks*sizeof(megablock),  num_blocks);


	mega_vqf * host_mega_vqf;

	mega_vqf * dev_mega_vqf;

	megablock * blocks;

	hipHostMalloc((void ** )& host_mega_vqf, sizeof(mega_vqf));

	hipMalloc((void ** )& dev_mega_vqf, sizeof(mega_vqf));	

	//init host
	host_mega_vqf->num_blocks = num_blocks;

	//allocate blocks
	hipMalloc((void **)&blocks, num_blocks*sizeof(megablock));

	hipMemset(blocks, 0, num_blocks*sizeof(megablock));

	host_mega_vqf->blocks = blocks;


	//external locks
	int * locks;
	hipMalloc((void ** )&locks, num_blocks*sizeof(int));
	hipMemset(locks, 0, num_blocks*sizeof(int));


	host_mega_vqf->locks = locks;



	hipMemcpy(dev_mega_vqf, host_mega_vqf, sizeof(mega_vqf), hipMemcpyHostToDevice);

	hipHostFree(host_mega_vqf);

	mega_vqf_block_setup<<<(num_blocks - 1)/64 + 1, 64>>>(dev_mega_vqf);
	hipDeviceSynchronize();

	return dev_mega_vqf;


}

#endif

