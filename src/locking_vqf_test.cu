#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


#include "include/vqf.cuh"

#include <openssl/rand.h>


#define BLOCK_SIZE 32

__global__ void test_insert_kernel(vqf* my_vqf, uint64_t nvals){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	//if (tid > 0) return;
	if (tid >= nvals) return;

	for (int i=0; i< 100; i++){

		my_vqf->insert(i);

	}

	// //does a single thread have this issue?
	// for (uint64_t i =0; i< nvals; i++){

	// 	assert(vals[i] != 0);

	// 	my_vqf->insert(vals[i]);

	// }

	
}


int main(int argc, char** argv) {
	




	uint64_t nitems = 28;

	

	vqf * my_vqf =  build_vqf(1);

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	test_insert_kernel<<<(nitems -1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(my_vqf, nitems);


	hipDeviceSynchronize();
	//and insert

	test_insert_kernel<<<(nitems -1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(my_vqf, nitems);


	hipDeviceSynchronize();

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Inserted " << nitems << " in " << diff.count() << " seconds\n";

  	printf("Inserts per second: %f\n", nitems/diff.count());





	

	return 0;

}
