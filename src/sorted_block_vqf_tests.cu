#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


#include "include/sorted_block_vqf.cuh"
#include "include/metadata.cuh"

#include <openssl/rand.h>






__host__ void insert_timing(optimized_vqf * my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	auto start = std::chrono::high_resolution_clock::now();


	my_vqf->sorted_bulk_insert(vals, nvals, misses);
	

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Inserted " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Inserts per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}


__global__ void test_query_kernel(optimized_vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;


	uint64_t teamID = tid / 32;
	int warpID = tid % 32;

	//if (tid > 0) return;
	if (teamID >= nvals) return;




	if(!my_vqf->query(warpID, vals[teamID])){

		my_vqf->query(warpID, vals[teamID]);

		if (warpID == 0)
		atomicAdd( (unsigned long long int *) misses, 1);
	}



	//printf("tid %llu done\n", tid);

	// //does a single thread have this issue?
	// for (uint64_t i =0; i< nvals; i++){

	// 	assert(vals[i] != 0);

	// 	my_vqf->insert(vals[i]);

	// }
	
}


__global__ void test_full_query_kernel(optimized_vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;


	uint64_t teamID = tid / 32;
	int warpID = tid % 32;

	//if (tid > 0) return;
	if (teamID >= nvals) return;




	if(!my_vqf->full_query(warpID, vals[teamID])){

		my_vqf->full_query(warpID, vals[teamID]);

		if (warpID == 0)
		atomicAdd( (unsigned long long int *) misses, 1);
	}



	//printf("tid %llu done\n", tid);

	// //does a single thread have this issue?
	// for (uint64_t i =0; i< nvals; i++){

	// 	assert(vals[i] != 0);

	// 	my_vqf->insert(vals[i]);

	// }
	
}

__host__ void query_timing(optimized_vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	auto start = std::chrono::high_resolution_clock::now();


	test_query_kernel<<<(32*nvals -1) / (32 * WARPS_PER_BLOCK) + 1, (32 * WARPS_PER_BLOCK)>>>(my_vqf, vals, nvals, misses);


	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}


__host__ void full_query_timing(optimized_vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	auto start = std::chrono::high_resolution_clock::now();


	test_full_query_kernel<<<(32*nvals -1) / (32 * WARPS_PER_BLOCK) + 1, (32 * WARPS_PER_BLOCK)>>>(my_vqf, vals, nvals, misses);


	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Full Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}


__host__ void sort_timing(optimized_vqf * my_vqf){


	auto start = std::chrono::high_resolution_clock::now();


	my_vqf->sort_and_check();

	hipDeviceSynchronize();

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Sorted in " << diff.count() << " seconds\n";


  	return;


}

__global__ void check_hits(bool * hits, uint64_t * misses, uint64_t nitems){


	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid >= nitems) return;

	if (!hits[tid]){

		atomicAdd((unsigned long long int *) misses, 1ULL);

	}
}

__host__ void bulk_query_timing(optimized_vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){



	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	

	my_vqf->bulk_query(vals, nvals, hits);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Bulk Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}


__host__ void sorted_bulk_query_timing(optimized_vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){



	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	

	my_vqf->sorted_bulk_query(vals, nvals, hits);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Sorted Bulk Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);  

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}



__host__ uint64_t * generate_data(uint64_t nitems){


	//malloc space

	uint64_t * vals = (uint64_t *) malloc(nitems * sizeof(uint64_t));


	//			   100,000,000
	uint64_t cap = 100000000ULL;

	for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

		uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


		RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(uint64_t));



		to_fill += togen;

		printf("Generated %llu/%llu\n", to_fill, nitems);

	}

	return vals;
}

int main(int argc, char** argv) {
	

	uint64_t nbits = atoi(argv[1]);


	uint64_t nitems = (1ULL << nbits) * .9;

	uint64_t * vals;
	uint64_t * dev_vals;

	uint64_t * other_vals;
	uint64_t * dev_other_vals;


	vals = generate_data(nitems);

	// vals = (uint64_t*) malloc(nitems*sizeof(vals[0]));

	// RAND_bytes((unsigned char *)vals, sizeof(*vals) * nitems);


	// other_vals = (uint64_t*) malloc(nitems*sizeof(other_vals[0]));

	// RAND_bytes((unsigned char *)other_vals, sizeof(*other_vals) * nitems);




	hipMalloc((void ** )& dev_vals, nitems*sizeof(vals[0]));

	hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);


	bool * inserts;


	hipMalloc((void ** )& inserts, nitems*sizeof(bool));

	hipMemset(inserts, 0, nitems*sizeof(bool));



	// hipMalloc((void ** )& dev_other_vals, nitems*sizeof(other_vals[0]));

	// hipMemcpy(dev_other_vals, other_vals, nitems * sizeof(other_vals[0]), hipMemcpyHostToDevice);


	//allocate misses counter
	uint64_t * misses;
	hipMallocManaged((void **)& misses, sizeof(uint64_t));

	misses[0] = 0;


	optimized_vqf * my_vqf =  build_vqf(1ULL << nbits);


	printf("Setup done\n");

	//wipe_vals<<<nitems/32+1, 32>>>(dev_vals, nitems);


	hipDeviceSynchronize();

	


	hipDeviceSynchronize();

	
	insert_timing(my_vqf, dev_vals, nitems,  misses);

	//hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	//sort_timing(my_vqf);

	hipDeviceSynchronize();

	//query_timing(my_vqf, dev_vals, nitems,  misses);

    hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);

// 	full_query_timing(my_vqf, dev_vals, nitems, misses);

	hipDeviceSynchronize();


// 	hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);


	sorted_bulk_query_timing(my_vqf, dev_vals, nitems, misses);

	hipDeviceSynchronize();
	


// 	bulk_query_timing(my_vqf, dev_vals, nitems, misses);

// 	// hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);

// 	// hipDeviceSynchronize();


// 	//remove_timing(my_vqf, dev_vals, inserts, nitems,  misses);



// 	hipDeviceSynchronize();

// //	sort_timing(my_vqf);

// 	hipDeviceSynchronize();


// 	hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);

// 	hipDeviceSynchronize();

// 	bulk_query_timing(my_vqf, dev_vals, nitems, misses);

// 	// hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);

// 	// hipDeviceSynchronize();


// 	//remove_timing(my_vqf, dev_vals, inserts, nitems,  misses);

// 	hipDeviceSynchronize();
// 	//and insert

// 	//auto end = std::chrono::high_resolution_clock::now();


	free(vals);

	hipFree(dev_vals);

	hipFree(misses);

	

	return 0;

}
