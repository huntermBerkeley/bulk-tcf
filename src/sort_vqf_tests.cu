#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


#include "include/team_vqf.cuh"

#include <openssl/rand.h>


//included thrust items for sorting
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>


#define BLOCK_SIZE 1024

#define GROUP_SIZE 8


 __host__ void sort_device_vector(uint64_t * vals, uint64_t nvals){


 	thrust::sort(thrust::device, vals, vals+nvals);

 	hipDeviceSynchronize();
 }

__global__ void test_insert_kernel(vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;


	uint64_t teamID = tid / 32;
	int warpID = tid % 32;

	//if (tid > 0) return;
	if (teamID*GROUP_SIZE >= nvals) return;



	for (uint64_t i = teamID*GROUP_SIZE; i < (teamID+1)*GROUP_SIZE; i++){

		if (i >= nvals) break;

		if (!my_vqf->insert(warpID, vals[i])){



		if (warpID == 0)
		atomicAdd( (unsigned long long int *) misses, 1);

		}


	}





	//printf("tid %llu done\n", tid);

	// //does a single thread have this issue?
	// for (uint64_t i =0; i< nvals; i++){

	// 	assert(vals[i] != 0);

	// 	my_vqf->insert(vals[i]);

	// }
	
}


__global__ void test_query_kernel(vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;


	uint64_t teamID = tid / 32;
	int warpID = tid % 32;



	if (teamID*GROUP_SIZE >= nvals) return;



	for (uint64_t i = teamID*GROUP_SIZE; i < (teamID+1)*GROUP_SIZE; i++){

		if (i >= nvals) break;

		if (!my_vqf->query(warpID, vals[i])){



		if (warpID == 0)
		atomicAdd( (unsigned long long int *) misses, 1);

		}


	}


	
}


__global__ void test_remove_kernel(vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;


	uint64_t teamID = tid / 32;
	int warpID = tid % 32;

	//if (tid > 0) return;
	if (teamID*GROUP_SIZE >= nvals) return;



	for (uint64_t i = teamID*GROUP_SIZE; i < (teamID+1)*GROUP_SIZE; i++){

		if (i >= nvals) break;

		if (!my_vqf->remove(warpID, vals[i])){



		if (warpID == 0)
		atomicAdd( (unsigned long long int *) misses, 1);

		}


	}



}



__host__ void insert_timing(vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	auto start = std::chrono::high_resolution_clock::now();


	test_insert_kernel<<<(32*nvals -1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(my_vqf, vals, nvals, misses);


	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Inserted " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Inserts per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}

__host__ void query_timing(vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	auto start = std::chrono::high_resolution_clock::now();


	test_query_kernel<<<(32*nvals -1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(my_vqf, vals, nvals, misses);


	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}


__host__ void remove_timing(vqf* my_vqf, uint64_t * vals, uint64_t nvals, uint64_t * misses){

	auto start = std::chrono::high_resolution_clock::now();


	test_remove_kernel<<<(32*nvals -1) / BLOCK_SIZE + 1, BLOCK_SIZE>>>(my_vqf, vals, nvals, misses);


	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


  	std::chrono::duration<double> diff = end-start;


  	std::cout << "removed " << nvals << " in " << diff.count() << " seconds\n";

  	printf("removes per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();
}

int main(int argc, char** argv) {
	

	uint64_t nbits = atoi(argv[1]);


	uint64_t nitems = (1 << nbits) * .8;

	uint64_t * vals;
	uint64_t * dev_vals;

	uint64_t * other_vals;
	uint64_t * dev_other_vals;

	vals = (uint64_t*) malloc(nitems*sizeof(vals[0]));

	RAND_bytes((unsigned char *)vals, sizeof(*vals) * nitems);


	// other_vals = (uint64_t*) malloc(nitems*sizeof(other_vals[0]));

	// RAND_bytes((unsigned char *)other_vals, sizeof(*other_vals) * nitems);




	hipMalloc((void ** )& dev_vals, nitems*sizeof(vals[0]));

	hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);


	// hipMalloc((void ** )& dev_other_vals, nitems*sizeof(other_vals[0]));

	// hipMemcpy(dev_other_vals, other_vals, nitems * sizeof(other_vals[0]), hipMemcpyHostToDevice);


	//allocate misses counter
	uint64_t * misses;
	hipMallocManaged((void **)& misses, sizeof(uint64_t));

	misses[0] = 0;


	vqf * my_vqf =  build_vqf(1 << nbits);


	sort_device_vector(dev_vals, nitems);


	printf("Setup done\n");

	hipDeviceSynchronize();

	
	insert_timing(my_vqf, dev_vals, nitems,  misses);

	query_timing(my_vqf, dev_vals, nitems,  misses);

	remove_timing(my_vqf, dev_vals, nitems,  misses);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


	free(vals);

	hipFree(dev_vals);

	hipFree(misses);

	

	return 0;

}
