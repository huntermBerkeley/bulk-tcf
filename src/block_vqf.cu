#include "hip/hip_runtime.h"

#ifndef OPTIMIZED_VQF_C
#define OPTIMIZED_VQF_C


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#include "include/block_vqf.cuh"
#include "include/gpu_block.cuh"
#include "include/hashutil.cuh"
#include "include/metadata.cuh"

#include <iostream>

#include <fstream>
#include <assert.h>

//Thrust Sorting
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>

#include <chrono>
#include <iostream>

#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;



struct is_tombstone
{
	__host__ __device__ bool operator()(const uint64_t val){

		return val == TOMBSTONE;
	}
};


__device__ void optimized_vqf::lock_block(int warpID, uint64_t team, uint64_t lock){


	// if (warpID == 0){

	// 	while(atomicCAS(locks + lock, 0,1) != 0);	
	// }
	// __syncwarp();

	//TODO: turn me back on

	#if EXCLUSIVE_ACCESS


	#else 
	blocks[team].internal_blocks[lock].lock(warpID);

	#endif
}

__device__ void optimized_vqf::unlock_block(int warpID, uint64_t team,  uint64_t lock){


	// if (warpID == 0){

	// 	while(atomicCAS(locks + lock, 1,0) != 1);	

	// }

	// __syncwarp();

	#if EXCLUSIVE_ACCESS


	#else

	blocks[team].internal_blocks[lock].unlock(warpID);

	#endif


}

__device__ void optimized_vqf::lock_blocks(int warpID, uint64_t team1, uint64_t lock1, uint64_t team2, uint64_t lock2){


	if (team1 * WARPS_PER_BLOCK + lock1 < team2 * WARPS_PER_BLOCK + lock2){

		lock_block(warpID, team1, lock1);
		lock_block(warpID, team2, lock2);
		//while(atomicCAS(locks + lock2, 0,1) == 1);

	} else {


		lock_block(warpID, team2, lock2);
		lock_block(warpID, team1, lock1);
		
	}

	


}

__device__ void optimized_vqf::unlock_blocks(int warpID, uint64_t team1, uint64_t lock1, uint64_t team2, uint64_t lock2){


	if (team1 * WARPS_PER_BLOCK + lock1 < team2 * WARPS_PER_BLOCK + lock2){

		unlock_block(warpID, team1, lock1);
		unlock_block(warpID, team2, lock2);
		//while(atomicCAS(locks + lock2, 0,1) == 1);

	} else {


		unlock_block(warpID, team2, lock2);
		unlock_block(warpID, team1, lock1);
		
	}

	


}

// __device__ bool optimized_vqf::insert(int warpID, uint64_t key, bool hashed){


// 	uint64_t hash;

// 	if (hashed){

// 		hash = key;

// 	} else {

// 		hash = hash_key(key);


// 	}

//    uint64_t block_index = get_bucket_from_hash(hash);

//    //uint64_t alt_block_index = get_alt_hash(hash, block_index);



//  	lock_block(warpID, block_index);

//    int fill_main = blocks[block_index].get_fill();



//    bool toReturn = false;


//    	if (fill_main < MAX_FILL){
//    		blocks[block_index].insert(warpID, hash);

//    		toReturn = true;


//    		#if DEBUG_ASSERTS
//    		int new_fill = blocks[block_index].get_fill();
//    		if (new_fill != fill_main+1){

//    		//blocks[block_index].printMetadata();
//    		printf("Broken Fill: Block %llu, old %d new %d\n", block_index, fill_main, new_fill);
//    		assert(blocks[block_index].get_fill() == fill_main+1);
//    		}

//    		assert(blocks[block_index].query(warpID, hash));
//    		#endif

//    	}

//    unlock_block(warpID, block_index);



//    return toReturn;





// }


//global call to create thread groups and trigger inserts;
//this is done inside of block_vqf.cu so that cg only needs to be brought in once
__global__ void bulk_insert_kernel(optimized_vqf * vqf){

	uint64_t tid = threadIdx.x + blockIdx.x*blockDim.x;


	//DEBUGGING - This was too small - resulted in multiple threads dropping early
	//nt sure if this is the only bug
	uint64_t teamID = tid / (BLOCK_SIZE);



	//TODO double check me
	if (teamID >= vqf->num_teams) return;


	vqf->mini_filter_insert(teamID);

	return;

	


}




//attach buffers, create thread groups, and launch
__host__ void optimized_vqf::bulk_insert(uint64_t * items, uint64_t nitems){



	uint64_t num_teams = get_num_teams();


	attach_buffers(items, nitems);

	bulk_insert_kernel<<<num_teams, BLOCK_SIZE>>>(this);



}


//once this is done TODO: add shared memory component
__device__ bool optimized_vqf::mini_filter_insert(uint64_t teamID){

	__shared__ thread_team_block block;

	//block = blocks[teamID];

	thread_block g = this_thread_block();

	//partition for first phase of buffered inserts

	const int subdivision_size = 32;


	//tile partition not splitting into 32?
	thread_block_tile<32> tile32 = tiled_partition<32>(g);

	int meta_rank = tile32.meta_group_rank();

	block.internal_blocks[meta_rank] = blocks[teamID].internal_blocks[meta_rank];
	//ew
	// for (uint64_t i = g.thread_rank() / subdivision_size; i < WARPS_PER_BLOCK; i+= g.size()/subdivision_size){

	// 		insert_single_buffer(tile32, teamID,  i);

	// }

	//replace this later, this is just to verify

	insert_single_buffer(tile32, &block, teamID, tile32.meta_group_rank());



	g.sync();




	//__syncthreads();




	blocks[teamID].internal_blocks[meta_rank] = block.internal_blocks[meta_rank];

	//after main inserts, the group will perform power of two inserts here

	//reserve a block at the end for inserts?
	//shuffle dump all blocks there


	return true;

}


__device__ bool optimized_vqf::insert_single_buffer(thread_block_tile<32> warpGroup, thread_team_block * local_blocks, uint64_t teamID, uint64_t buffer){



	#if DEBUG_ASSERTS

	assert(teamID  < num_teams);

	assert( (teamID * WARPS_PER_BLOCK + buffer) < num_blocks);

	#endif

	//at this point the team should be referring to a valid target for insertions
	//this is a copy of buffer_insert modified to the use the cooperative group API
	//for the original version check optimized_vqf.cu::buffer_insert

	//local_blocks->internal_blocks[buffer];

	uint64_t global_buffer = teamID*WARPS_PER_BLOCK + buffer;


	int count = FILL_CUTOFF - local_blocks->internal_blocks[buffer].get_fill();

	int buf_size = buffer_sizes[global_buffer];

	if (buf_size < count) count = buf_size;


	// if (warpGroup.thread_rank() != 0){
	// 	printf("Halp: %d %d\n", warpGroup.thread_rank(), count);
	// }

	//modify to be warp group specific

	local_blocks->internal_blocks[buffer].bulk_insert_team(warpGroup, buffers[global_buffer], count);


	//block.bulk_insert(warpGroup.thread_rank(), buffers[global_buffer], count);


	//local_blocks->internal_blocks[buffer] = block;

	if (warpGroup.thread_rank() == 0){

		buffers[global_buffer] += count;

		buffer_sizes[global_buffer] -= count;
	}

}



// __device__ bool optimized_vqf::finalize_thread_group(thread_group teamGroup, uint64_t teamID){



// 	//starting with teamGroup and threadGroup - this is just gonna go after the main code


// }


// __device__ void optimized_vqf::dump_thread_group_reserved(thread_group teamGroup, uint64_t teamID){


// 	//thread group local atomic? that would be great


// }


// __device__ int optimized_vqf::buffer_query(int warpID, uint64_t buffer){


// 	#if DEBUG_ASSERTS

// 	assert(buffer < num_blocks);

// 	#endif


// 	uint64_t block_index = buffer;

// 	lock_block(warpID, block_index);

	
// 	int buf_size = buffer_sizes[buffer];


// 	int found = blocks[block_index].bulk_query(warpID, buffers[buffer], buf_size);
	

// 	unlock_block(warpID, block_index);

// 	//and decrement the count



// 	return buf_size - found;


// }




// __device__ bool vqf::shared_buffer_insert(int warpID, int shared_blockID, uint64_t buffer){


// 	__shared__ vqf_block extern_blocks[WARPS_PER_BLOCK];


// 	#if DEBUG_ASSERTS

// 	assert(buffer < num_blocks);

// 	#endif


// 	uint64_t block_index = buffer;

// 	//lock_block(warpID, block_index);


// 	if (warpID == 0) extern_blocks[shared_blockID] = blocks[block_index];

// 	//extern_blocks[shared_blockID].load_block(warpID, blocks + block_index);

// 	extern_blocks[shared_blockID].lock(warpID);


// 	int fill_main = extern_blocks[shared_blockID].get_fill();

// 	#ifdef DEBUG_ASSERTS
// 	assert(fill_main == 0);
// 	#endif

// 	int count = FILL_CUTOFF - fill_main;

// 	int buf_size = buffer_sizes[buffer];

// 	if (buf_size < count) count = buf_size;

// 	for (int i =0; i < count; i++){

// 		#if DEBUG_ASSERTS

// 		int old_fill = extern_blocks[shared_blockID].get_fill();


// 		//relevant equation

// 		// (x mod yz) | z == x mod y?
// 		//python says no ur a dumbass this is the bug
		
// 		if (!(get_bucket_from_hash(buffers[buffer][i])  == buffer)){

// 			if (warpID == 0){

// 				printf("i %d count %d item %llu buffer %llu new_buf %llu\n", i, count, buffers[buffer][i], buffer, get_bucket_from_hash(buffers[buffer][i]));
// 			}

// 			__syncwarp();

// 			assert((buffers[buffer][i] >> TAG_BITS) % num_blocks  == buffer);

// 		}
		


// 		#endif

// 		uint64_t tag = buffers[buffer][i] & ((1ULL << TAG_BITS) -1);
// 		extern_blocks[shared_blockID].insert(warpID, tag);

// 		#if DEBUG_ASSERTS

// 		assert(extern_blocks[shared_blockID].get_fill() == old_fill+1);

// 		#endif

// 	}

// 	//write back

// 	extern_blocks[shared_blockID].unlock(warpID);


// 	//if (warpID == 0)
// 	//blocks[block_index] = extern_blocks[shared_blockID];

// 	//blocks[block_index].load_block(warpID, extern_blocks + shared_blockID);


// 	__threadfence();
// 	__syncwarp();

// 	//blocks[block_index].unlock(warpID);
	

// 	//and decrement the count

// 	if (warpID == 0){

// 		buffers[buffer] += count;

// 		buffer_sizes[buffer] -= count;


// 	}


// }


// __device__ bool vqf::multi_buffer_insert(int warpID, int init_blockID, uint64_t start_buffer){


// 	__shared__ vqf_block extern_blocks[WARPS_PER_BLOCK*REGIONS_PER_WARP];


// 	#if DEBUG_ASSERTS

// 	assert(start_buffer < num_blocks);

// 	#endif


// 	int shared_blockID = init_blockID * REGIONS_PER_WARP;



// 	if (start_buffer + warpID < num_blocks)

// 	{


// 		extern_blocks[shared_blockID + warpID % REGIONS_PER_WARP] = blocks[start_buffer + warpID % REGIONS_PER_WARP];

// 	}

// 	__syncwarp();

// 	for (int i = 0; i < REGIONS_PER_WARP; i++){

// 		if (start_buffer + i >= num_blocks) break;

// 		extern_blocks[shared_blockID + i].lock(warpID);
// 	}


// 	// 	


// 	// }

// 	__syncwarp();
	

// 	for (int i = 0; i < REGIONS_PER_WARP; i++){

// 		if (start_buffer + i >= num_blocks) break;

// 		int extern_id = shared_blockID + i;

// 		uint64_t buffer = start_buffer + i;



// 		int fill_main = extern_blocks[extern_id].get_fill();

// 		#ifdef DEBUG_ASSERTS
// 		assert(fill_main == 0);
// 		#endif

// 		int count = FILL_CUTOFF - fill_main;

// 		int buf_size = buffer_sizes[buffer];

// 		if (buf_size < count) count = buf_size;

// 		for (int i =0; i < count; i++){




// 			#if DEBUG_ASSERTS

// 			int old_fill = extern_blocks[extern_id].get_fill();


// 			//relevant equation

// 			// (x mod yz) | z == x mod y?
// 			//python says no ur a dumbass this is the bug
			
// 			if (!(get_bucket_from_hash(buffers[buffer][i])  == buffer)){

// 				if (warpID == 0){

// 					printf("i %d count %d item %llu buffer %llu new_buf %llu\n", i, count, buffers[buffer][i], buffer, get_bucket_from_hash(buffers[buffer][i]));
// 				}

// 				__syncwarp();

// 				assert((buffers[buffer][i] >> TAG_BITS) % num_blocks  == buffer);

// 			}
			


// 			#endif

// 			uint64_t tag = buffers[buffer][i] & ((1ULL << TAG_BITS) -1);
// 			extern_blocks[extern_id].insert(warpID, tag);

// 			#if DEBUG_ASSERTS

// 			assert(extern_blocks[extern_id].get_fill() == old_fill+1);

// 			#endif

// 		}


// 	//wrap up the loops

// 	extern_blocks[extern_id].unlock(warpID);

// 	if (warpID == 0){

// 		buffers[buffer] += count;

// 		buffer_sizes[buffer] -= count;


// 	}

// 	__syncwarp();

// 	}




// 	//write back

// 	for (int i = 0; i < REGIONS_PER_WARP; i++){

// 		if (start_buffer + i >= num_blocks) break;

		
// 		extern_blocks[shared_blockID + i].unlock(warpID);
// 	}


// 		if (start_buffer + warpID < num_blocks)

// 			{


// 			blocks[start_buffer + warpID % REGIONS_PER_WARP] = extern_blocks[shared_blockID + warpID % REGIONS_PER_WARP];


// 		}
// 	//if (warpID == 0)
// 	//blocks[block_index] = extern_blocks[shared_blockID];

// 	//blocks[block_index].load_block(warpID, extern_blocks + shared_blockID);


// 	__threadfence();
// 	__syncwarp();

// 	//blocks[block_index].unlock(warpID);
	

// 	//and decrement the count



// }

//Double check that the two inserts line up!
//to activate, tab out the code that changes the sizes of the buffers in buffer_insert
//otherwise results get wacky
// __device__ bool vqf::shared_buffer_insert_check(int warpID, int shared_blockID, uint64_t buffer){



// 	__shared__ vqf_block extern_blocks[WARPS_PER_BLOCK];

// 	#if DEBUG_ASSERTS

// 	assert(buffer < num_blocks);

// 	#endif


// 	uint64_t block_index = buffer;

// 	lock_block(warpID, block_index);


// 	if (warpID == 0)

// 	extern_blocks[shared_blockID] = blocks[block_index];


// 	if (!compare_blocks(blocks[block_index],extern_blocks[shared_blockID])){

// 		assert(compare_blocks(blocks[block_index],extern_blocks[shared_blockID]));

// 	}


// 	int fill_main = extern_blocks[shared_blockID].get_fill();

// 	#ifdef DEBUG_ASSERTS
// 	assert(fill_main == 0);
// 	#endif

// 	int count = FILL_CUTOFF - fill_main;

// 	int buf_size = buffer_sizes[buffer];

// 	if (buf_size < count) count = buf_size;

// 	for (int i =0; i < count; i++){

// 		#if DEBUG_ASSERTS

// 		int old_fill = extern_blocks[shared_blockID].get_fill();


// 		//relevant equation

// 		// (x mod yz) | z == x mod y?
// 		//python says no ur a dumbass this is the bug
		
// 		if (!(get_bucket_from_hash(buffers[buffer][i])  == buffer)){

// 			if (warpID == 0){

// 				printf("i %d count %d item %llu buffer %llu new_buf %llu\n", i, count, buffers[buffer][i], buffer, get_bucket_from_hash(buffers[buffer][i]));
// 			}

// 			__syncwarp();

// 			assert((buffers[buffer][i] >> TAG_BITS) % num_blocks  == buffer);

// 		}
		


// 		#endif

// 		uint64_t tag = buffers[buffer][i] & ((1ULL << TAG_BITS) -1);

// 		blocks[block_index].insert(warpID, tag);
// 		extern_blocks[shared_blockID].insert(warpID, tag);

// 		#if DEBUG_ASSERTS


// 		if (!compare_blocks(blocks[block_index],extern_blocks[shared_blockID])){

// 			assert(compare_blocks(blocks[block_index],extern_blocks[shared_blockID]));

// 		}
		

// 		assert(extern_blocks[shared_blockID].get_fill() == old_fill+1);

// 		assert(blocks[block_index].get_fill() == old_fill + 1);

// 		#endif

// 	}

// 	//write back

// 	if (!compare_blocks(blocks[block_index],extern_blocks[shared_blockID])){

// 		assert(compare_blocks(blocks[block_index],extern_blocks[shared_blockID]));

// 	}

// 	__threadfence();
// 	__syncwarp();

// 	blocks[block_index].unlock(warpID);

// 	//and decrement the count

// 	if (warpID == 0){

// 		buffers[buffer] += count;

// 		buffer_sizes[buffer] -= count;


// 	}


// }


//come back and put me in the final implementation
// __device__ bool vqf::buffer_end_dump(int warpID, uint64_t buffer){


// 	int count = buffer_sizes[buffer];

// 	for (int i =0; i < )
// }


__device__ bool optimized_vqf::query(int warpID, uint64_t key){

	uint64_t hash = hash_key(key);

	//uint64_t block_index = ((hash >> TAG_BITS) % (VIRTUAL_BUCKETS*num_blocks))/VIRTUAL_BUCKETS;
	uint64_t block_index = get_bucket_from_hash(hash);

   //this will generate a mask and get the tag bits
   //uint64_t tag = hash & ((1ULL << TAG_BITS) -1);
   //uint64_t alt_block_index = (((hash ^ (tag * 0x5bd1e995)) % (num_blocks*SLOTS_PER_BLOCK)) >> TAG_BITS) % num_blocks;
	//uint64_t alt_block_index = get_alt_hash(hash, block_index);

  //  while (block_index == alt_block_index){
		// alt_block_index = (alt_block_index * (tag * 0x5bd1e995)) % num_blocks;
  //  }


   uint64_t team_index = block_index / WARPS_PER_BLOCK;

   block_index = block_index % WARPS_PER_BLOCK;


   lock_block(warpID, team_index, block_index);

   #if DEBUG_ASSERTS
 	assert(blocks[team_index].internal_blocks[block_index].assert_consistency());

 	#endif
   bool found = blocks[team_index].internal_blocks[block_index].query(warpID, hash);

   #if DEBUG_ASSERTS
   assert(blocks[team_index].internal_blocks[block_index].assert_consistency());
   #endif

  	unlock_block(warpID, team_index, block_index);

   return found;

}

// __device__ bool optimized_vqf::full_query(int warpID, uint64_t key){

// 	uint64_t hash = hash_key(key);

// 	//uint64_t block_index = ((hash >> TAG_BITS) % (VIRTUAL_BUCKETS*num_blocks))/VIRTUAL_BUCKETS;
// 	uint64_t block_index = get_bucket_from_hash(hash);

//    //this will generate a mask and get the tag bits
//    //uint64_t tag = hash & ((1ULL << TAG_BITS) -1);
//    //uint64_t alt_block_index = (((hash ^ (tag * 0x5bd1e995)) % (num_blocks*SLOTS_PER_BLOCK)) >> TAG_BITS) % num_blocks;
// 	//uint64_t alt_block_index = get_alt_hash(hash, block_index);

//   //  while (block_index == alt_block_index){
// 		// alt_block_index = (alt_block_index * (tag * 0x5bd1e995)) % num_blocks;
//   //  }



//    lock_block(warpID, block_index);

//    #if DEBUG_ASSERTS
//  	assert(blocks[block_index].assert_consistency());

//  	#endif
//    bool found = blocks[block_index].query(warpID, hash);

//    #if DEBUG_ASSERTS
//    assert(blocks[block_index].assert_consistency());
//    #endif

//   	unlock_block(warpID, block_index);



//   	if (found) return true;

//    //check the other block

//   	uint64_t alt_hash = get_alt_hash(hash, block_index);

//    uint64_t alt_block_index = get_bucket_from_hash(alt_hash);

//    lock_block(warpID, alt_block_index);


//    found = blocks[alt_block_index].query(warpID, alt_hash);

//    unlock_block(warpID, alt_block_index);

//    return found;
// }


//BUG: insert and remove seems to not be correct
//V1: uint64_t block_index = ((hash >> TAG_BITS) % (VIRTUAL_BUCKETS*num_blocks))/VIRTUAL_BUCKETS;
//V2: uint64_t block_index = (hash >> TAG_BITS) % num_blocks;

// __device__ bool optimized_vqf::remove(int warpID, uint64_t key){

// 	uint64_t hash = hash_key(key);


// 	uint64_t block_index = get_bucket_from_hash(hash);

//    //this will generate a mask and get the tag bits
// 	//uint64_t alt_block_index = get_alt_hash(hash, block_index);

//   //  while (block_index == alt_block_index){
// 		// alt_block_index = (alt_block_index * (tag * 0x5bd1e995)) % num_blocks;
//   //  }

//   		lock_block(warpID, block_index);


//   		#if DEBUG_ASSERTS

//   		assert(blocks[block_index].assert_consistency());


// 		int old_fill = blocks[block_index].get_fill();

// 		//assert(blocks[block_index].assert_consistency());

// 		uint64_t md_before = blocks[block_index].md[0];


// 		#endif

//    bool found = blocks[block_index].remove(warpID, hash);


//       #if DEBUG_ASSERTS
//  		int new_fill = blocks[block_index].get_fill();

//  		//assert(blocks[block_index].assert_consistency());

//  		uint64_t md_after = blocks[block_index].md[0];

//  		if (!found){

//  			assert(md_before == md_after);

 			

//  		} else {

//  			assert(new_fill >= 0);

//  			if(old_fill-1 != new_fill){


//  				assert(blocks[block_index].assert_consistency());
//  				blocks[block_index].remove(warpID, hash);

//  				assert(old_fill-1 == new_fill);
//  			}
//  		}
 		

 		

//  		#endif

//    unlock_block(warpID, block_index);

//    //copy could be deleted from this instance

// 	 return found;

// }


// __device__ bool vqf::insert(uint64_t hash){

//    uint64_t block_index = (hash >> TAG_BITS) % num_blocks;



//    //this will generate a mask and get the tag bits
//    uint64_t tag = hash & ((1ULL << TAG_BITS) -1);
//    uint64_t alt_block_index = (((hash ^ (tag * 0x5bd1e995)) % (num_blocks*SLOTS_PER_BLOCK)) >> TAG_BITS) % num_blocks;

//    assert(block_index < num_blocks);


//    //external locks
//    //blocks[block_index].extra_lock(block_index);
   
//    while(atomicCAS(locks + block_index, 0, 1) == 1);



//    int fill_main = blocks[block_index].get_fill();


//    if (fill_main >= SLOTS_PER_BLOCK-1){

//    	while(atomicCAS(locks + block_index, 0, 1) == 0);
//    	//blocks[block_index].unlock();

//    	return false;
//    }

//    if (fill_main < .75 * SLOTS_PER_BLOCK || block_index == alt_block_index){
//    	blocks[block_index].insert(tag);

   	

//    	int new_fill = blocks[block_index].get_fill();
//    	if (new_fill != fill_main+1){
//    		printf("Broken Fill: Block %llu, old %d new %d\n", block_index, fill_main, new_fill);
//    		assert(blocks[block_index].get_fill() == fill_main+1);
//    	}


//    	while(atomicCAS(locks + block_index, 1, 0) == 0);
//    	//blocks[block_index].unlock();
//    	return true;
//    }


//    while(atomicCAS(locks + block_index, 1, 0) == 0);

//    lock_blocks(block_index, alt_block_index);


//    //need to grab other block

//    //blocks[alt_block_index].extra_lock(alt_block_index);
//    while(atomicCAS(locks + alt_block_index, 0, 1) == 1);

//    int fill_alt = blocks[alt_block_index].get_fill();

//    //any larger and we can't protect metadata
//    if (fill_alt >=  SLOTS_PER_BLOCK-1){
// //   	blocks[block_index.unlock()]

//    	unlock_blocks(block_index, alt_block_index);
//    	//blocks[alt_block_index].unlock();
//    	//blocks[block_index].unlock();
//    	return false;
//    }


//    //unlock main
//    if (fill_main > fill_alt ){

//    	while(atomicCAS(locks + block_index, 1, 0) == 0);
//    	//blocks[block_index].unlock();

//    	blocks[alt_block_index].insert(tag);
//    	assert(blocks[alt_block_index].get_fill() == fill_alt+1);

//    	int new_fill = blocks[alt_block_index].get_fill();
//    	if (new_fill != fill_alt+1){
//    		printf("Broken Fill: Block %llu, old %d new %d\n", alt_block_index, fill_alt, new_fill);
//    		assert(blocks[alt_block_index].get_fill() == fill_alt+1);
//    	}

//    	while(atomicCAS(locks + alt_block_index, 1, 0) == 0);
//    	//blocks[alt_block_index].unlock();


//    } else {

//    	while(atomicCAS(locks + alt_block_index, 1, 0) == 0);
//    	//blocks[alt_block_index].unlock();
//    	blocks[block_index].insert(tag);

//    	int new_fill = blocks[block_index].get_fill();
//    	if (new_fill != fill_main+1){
//    		printf("Broken Fill: Block %llu, old %d new %d\n", block_index, fill_main, new_fill);
//    		assert(blocks[block_index].get_fill() == fill_main+1);
//    	}

//    	while(atomicCAS(locks + block_index, 1, 0) == 0);
//    	//blocks[block_index].unlock();

//    }


  
//    return true;



//}

__device__ uint64_t optimized_vqf::hash_key(uint64_t key){


	key = MurmurHash64A(((void *)&key), sizeof(key), seed) % ((num_blocks * VIRTUAL_BUCKETS) << TAG_BITS);

	return key;


}

__global__ void hash_all(optimized_vqf* my_vqf, uint64_t* vals, uint64_t* hashes, uint64_t nvals) {
	
	uint64_t idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= nvals){
		return;
	}

  uint64_t key = vals[idx];

  key = my_vqf->hash_key(key);


	//uint64_t hash = (key << qf->metadata->value_bits) | (value & BITMASK(qf->metadata->value_bits));


  #if DEBUG_ASSERTS

  assert(key >> TAG_BITS < my_vqf->num_blocks* VIRTUAL_BUCKETS);

  #endif
  
  hashes[idx] = key;

	return;

}


//given a hashed list of inserts, convert the hashes into their alternate hash
// this is used to quickly convert the second half of the 
__global__ void alt_hash_all(optimized_vqf * my_vqf, uint64_t * vals, uint64_t nvals){



	uint64_t tid = threadIdx.x + blockDim.x * blockIdx.x;

	if (tid >= nvals) return;

	uint64_t key = vals[tid];

	key = my_vqf->get_alt_hash(key, my_vqf->get_bucket_from_hash(key));

	vals[tid] = key;
}


//set the references with tid[i] = i;
__global__ void init_references(uint64_t * vals, uint64_t nvals){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid >= 2*nvals) return;

	vals[tid] = tid;

}





//Bug thoughts

//keys run over range 0 - num_blocks*virtual_buckets >> tags

//to pick a slot, down shift tags to get 0-num_blocks*virtual_buckets

__global__ void set_buffers_binary(optimized_vqf * my_vqf, uint64_t num_keys, const __restrict__ uint64_t * keys){

		uint64_t idx = threadIdx.x + blockDim.x * blockIdx.x;

		if (idx >= my_vqf->num_blocks) return;

		//uint64_t slots_per_lock = VIRTUAL_BUCKETS;

		//since we are finding all boundaries, we only need

		//printf("idx %llu\n", idx);

		//this sounds right? - they divide to go back so I think this is fine
		//this is fine but need to apply a hash
		uint64_t boundary = idx; //<< qf->metadata->bits_per_slot;


		//This is the code I'm stealing that assumption from
		//uint64_t hash_bucket_index = hash >> qf->metadata->bits_per_slot;
		//uint64_t hash_remainder = hash & BITMASK(qf->metadata->bits_per_slot);	
		//uint64_t lock_index = hash_bucket_index / slots_per_lock;


		uint64_t lower = 0;
		uint64_t upper = num_keys;
		uint64_t index = upper-lower;

		//upper is non inclusive bound


		//if we exceed bounds that's our index
		while (upper != lower){


			index = lower + (upper - lower)/2;

			//((keys[index] >> TAG_BITS)
			uint64_t bucket = my_vqf->get_bucket_from_hash(keys[index]);


			if (index != 0)
			uint64_t old_bucket = my_vqf->get_bucket_from_hash(keys[index-1]);

			if (bucket < boundary){

				//false - the list before this point can be removed
				lower = index+1;

				//jump to a new midpoint
				


			} else if (index==0){

				//will this fix? otherwise need to patch via round up
				upper = index;

				//(get_bucket_from_hash(keys[index-1])
				//(keys[index-1] >> TAG_BITS)

			} else if (my_vqf->get_bucket_from_hash(keys[index-1]) < boundary) {

				//set index! this is the first instance where I am valid and the next isnt
				//buffers[idx] = keys+index;
				break;

			} else {

				//we are too far right, all keys to the right do not matter
				upper = index;


			}

		}

		//we either exited or have an edge condition:
		//upper == lower iff 0 or max key
		index = lower + (upper - lower)/2;

		assert(my_vqf->get_bucket_from_hash(keys[index]) == idx);


		my_vqf->buffers[idx] = ((uint64_t *)keys) + index;
		


}

//this can maybe be rolled into set_buffers_binary
//it performs an identical set of operations that are O(1) here
// O(log n) there, but maybe amortized

__global__ void set_buffer_lens(optimized_vqf* my_vqf, uint64_t num_keys, const __restrict__ uint64_t * keys){


	uint64_t num_buffers = my_vqf->num_blocks;


	uint64_t idx = threadIdx.x + blockDim.x*blockIdx.x;

	if (idx >= num_buffers) return;


	//only 1 thread will diverge - should be fine - any cost already exists because of tail
	if (idx != num_buffers-1){

		//this should work? not 100% convinced but it seems ok
		my_vqf->buffer_sizes[idx] = my_vqf->buffers[idx+1] - my_vqf->buffers[idx];
	} else {

		my_vqf->buffer_sizes[idx] = num_keys - (my_vqf->buffers[idx] - keys);

	}

	return;


}

__host__ uint64_t optimized_vqf::get_num_buffers(){

	uint64_t internal_num_blocks;

	hipMemcpy(&internal_num_blocks, (uint64_t * ) this, sizeof(uint64_t), hipMemcpyDeviceToHost);

 	hipDeviceSynchronize();

 	return internal_num_blocks;
}

__host__ uint64_t optimized_vqf::get_num_teams(){

	uint64_t internal_num_teams;

	hipMemcpy(&internal_num_teams, ((uint64_t * ) this) + 1, sizeof(uint64_t), hipMemcpyDeviceToHost);

 	hipDeviceSynchronize();

 	return internal_num_teams;
}


//have the VQF sort the input dataset and attach the buffers to the data

__host__ void optimized_vqf::attach_buffers(uint64_t * vals, uint64_t nvals){



	hash_all<<<(nvals - 1)/1024 + 1, 1024>>>(this, vals, vals, nvals);


	thrust::sort(thrust::device, vals, vals+nvals);




	uint64_t internal_num_blocks = get_num_buffers();
	


 	set_buffers_binary<<<(internal_num_blocks - 1)/1024 +1, 1024>>>(this, nvals, vals);

 	set_buffer_lens<<<(internal_num_blocks - 1)/1024 +1, 1024>>>(this, nvals, vals);


}


__global__ void vqf_block_setup(optimized_vqf * vqf){

	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid >= vqf->num_blocks) return;


 
	vqf->blocks[tid / WARPS_PER_BLOCK].internal_blocks[tid % WARPS_PER_BLOCK].setup();


	#if EXCLUSIVE_ACCESS

	vqf->blocks[tid / WARPS_PER_BLOCK].internal_blocks[tid % WARPS_PER_BLOCK].lock(0);

	#endif

}

__host__ optimized_vqf * build_vqf(uint64_t nitems){


	#if DEBUG_ASSERTS

	printf("Debug correctness checks on. These will affect performance.\n");

	#endif

	//this seems weird but whatever
	uint64_t num_blocks = (nitems -1)/SLOTS_PER_BLOCK + 1;

	uint64_t num_teams = (num_blocks-1) / WARPS_PER_BLOCK + 1;

	//rewrite num_blocks to account for any expansion.
	num_blocks = num_teams*WARPS_PER_BLOCK;

	printf("Bytes used: %llu for %llu blocks.\n", num_teams*sizeof(thread_team_block),  num_blocks);


	optimized_vqf * host_vqf;

	optimized_vqf * dev_vqf;

	thread_team_block * blocks;

	hipHostMalloc((void ** )& host_vqf, sizeof(optimized_vqf));

	hipMalloc((void ** )& dev_vqf, sizeof(optimized_vqf));	

	//init host
	host_vqf->num_blocks = num_blocks;

	host_vqf->num_teams = num_teams;

	//allocate blocks
	hipMalloc((void **)&blocks, num_teams*sizeof(thread_team_block));

	hipMemset(blocks, 0, num_teams*sizeof(thread_team_block));

	host_vqf->blocks = blocks;


	//external locks

	//TODO: get rid of these they're not necessary
	int * locks;

	//numblocks or 1
	hipMalloc((void ** )&locks,1*sizeof(int));
	hipMemset(locks, 0, 1*sizeof(int));


	host_vqf->locks = locks;


	uint64_t ** buffers;
	uint64_t * buffer_sizes;

	//in this scheme blocks are per 

	hipMalloc((void **)& buffers, num_blocks*sizeof(uint64_t *));
	hipMemset(buffers, 0, num_blocks*sizeof(uint64_t * ));

	hipMalloc((void **)& buffer_sizes, num_blocks*sizeof(uint64_t));
	hipMemset(buffer_sizes, 0, num_blocks*sizeof(uint64_t));


	host_vqf->buffers = buffers;

	host_vqf->buffer_sizes = buffer_sizes;

	host_vqf->seed = 5;


	hipMemcpy(dev_vqf, host_vqf, sizeof(optimized_vqf), hipMemcpyHostToDevice);

	hipHostFree(host_vqf);

	vqf_block_setup<<<(num_blocks - 1)/64 + 1, 64>>>(dev_vqf);
	hipDeviceSynchronize();

	return dev_vqf;


}


//the upper (sizeof(hash - TAG_BITS)) represent the slot of the hash,
// downshift and modulus to get the slot, and then divide to get the bucket the slot belongs to
//This is outdated but still works, a maintains a guarantee that items are assigned in sorted order
//which is a precondition for the bucket inserts and power of two inserts.
__device__ uint64_t optimized_vqf::get_bucket_from_hash(uint64_t hash){

	return ((hash >> TAG_BITS) % (num_blocks * VIRTUAL_BUCKETS)) / VIRTUAL_BUCKETS;
}



//TODO: modify this to only refer to the local buckets


//generate the alternate hash for inserts, the new version requires a call to 
// get_bucket+from_hash as well, but has the additional benefit of returning a working key.
__device__ uint64_t optimized_vqf::get_alt_hash(uint64_t hash, uint64_t bucket){

	uint64_t alt_block_index = hash_key(hash);


	// while (alt_block_index == bucket){
	// 	alt_block_index = get_bucket_from_hash(hash_key(hash ^ alt_block_index));
	// }

	//ask prashant for a better way to do this it feels ridiculous.
	while (get_bucket_from_hash(alt_block_index) == bucket){

	//I goofed here and some items stall if you add just 1
	//you should jump one bucket instead, SLOTS_PER_BLOCK << TAG_BITS
	alt_block_index = (alt_block_index + SLOTS_PER_BLOCK << TAG_BITS);

	}

	return alt_block_index;
}








#endif

