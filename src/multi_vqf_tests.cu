/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


#include "include/multi_vqf_host.cuh"
#include "include/metadata.cuh"

#include <openssl/rand.h>







int main(int argc, char** argv) {
	

	uint64_t nbits = atoi(argv[1]);


	uint64_t nitems = (1ULL << nbits) * .9;


	multi_vqf * m_vqf;

	m_vqf = build_vqf(10, nbits);

	hipDeviceSynchronize();



	return 0;

}
