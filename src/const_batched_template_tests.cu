#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>


#include "include/const_block_templated_vqf.cuh"
#include "include/metadata.cuh"

#include <openssl/rand.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void check_hits(bool * hits, uint64_t * misses, uint64_t nitems){


	uint64_t tid = threadIdx.x + blockDim.x*blockIdx.x;

	if (tid >= nitems) return;

	if (!hits[tid]){

		atomicAdd((unsigned long long int *) misses, 1ULL);

	}
}

template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> split_insert_timing(templated_vqf<Key, Val, Wrapper> * my_vqf, uint64_t * reference_vals, key_val_pair<Key, Val, Wrapper> * vals, uint64_t nvals, uint64_t * misses){


	uint64_t num_blocks = my_vqf->get_num_blocks();

	uint64_t num_teams = my_vqf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();

	my_vqf->attach_lossy_buffers(reference_vals, vals, nvals, num_blocks);


	hipDeviceSynchronize();
	
	gpuErrchk( hipPeekAtLastError() );


	auto midpoint = std::chrono::high_resolution_clock::now();


	my_vqf->bulk_insert(misses, num_teams);
	

	hipDeviceSynchronize();

	gpuErrchk( hipPeekAtLastError() );
	//and insert

	auto end = std::chrono::high_resolution_clock::now();


	std::chrono::duration<double> attach_diff = midpoint-start;
  	std::chrono::duration<double> insert_diff = end-midpoint;	
  	std::chrono::duration<double> diff = end-start;



  	std::cout << "attached in " << attach_diff.count() << ", inserted in " << insert_diff.count() << ".\n";

  	std::cout << "Inserted " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Inserts per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();

  	return diff;
}


template <typename Filter, typename Key_type>
__global__ void single_warp_insert_kernel(Filter * my_vqf, uint64_t * reference_vals, Key_type * vals, uint64_t nitems, uint64_t * misses){


	uint64_t tid = threadIdx.x+blockDim.x*blockIdx.x;

	uint64_t itemID = tid / 32;
	int warpID = tid % 32;

	if (itemID >= nitems) return;

	my_vqf->dump_item_into_block(reference_vals[itemID], vals[itemID], warpID, misses);


}


template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> single_insert_timing(templated_vqf<Key, Val, Wrapper> * my_vqf, uint64_t * reference_vals, key_val_pair<Key, Val, Wrapper> * vals, uint64_t nvals, uint64_t * misses){


	uint64_t num_blocks = my_vqf->get_num_blocks();

	uint64_t num_teams = my_vqf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();

	const int block_size = 512;

	single_warp_insert_kernel<templated_vqf<Key, Val, Wrapper>, key_val_pair<Key, Val, Wrapper>><<<(nvals*32-1)/block_size+1, block_size>>>(my_vqf, reference_vals, vals, nvals, misses);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits



  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Inserted " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Bulk Inserts per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();


  	return diff;
}


template <typename Filter, typename Key_type>
__global__ void single_warp_query_kernel(Filter * my_vqf, uint64_t * reference_vals, Key_type * vals, uint64_t nitems, bool * hits){


	uint64_t tid = threadIdx.x+blockDim.x*blockIdx.x;

	uint64_t itemID = tid / 32;
	int warpID = tid % 32;

	if (itemID >= nitems) return;

	hits[itemID] = my_vqf->query_single_item(reference_vals[itemID], vals[itemID], warpID);


}

template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> single_bulk_query_timing(templated_vqf<Key, Val, Wrapper> * my_vqf, uint64_t * reference_vals, key_val_pair<Key, Val, Wrapper> * vals, uint64_t nvals, uint64_t * misses){



	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	
	const int block_size = 512;

	single_warp_query_kernel<templated_vqf<Key, Val, Wrapper>, key_val_pair<Key, Val, Wrapper>><<<(nvals*32-1)/block_size+1, block_size>>>(my_vqf, reference_vals, vals, nvals, hits);


	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Bulk Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();


  	return diff;
}


template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> single_fp_timing(templated_vqf<Key, Val, Wrapper> * my_vqf, uint64_t * reference_vals, key_val_pair<Key, Val, Wrapper> * vals, uint64_t nvals, uint64_t * misses){



	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	
	const int block_size = 512;

	single_warp_query_kernel<templated_vqf<Key, Val, Wrapper>, key_val_pair<Key, Val, Wrapper>><<<(nvals*32-1)/block_size+1, block_size>>>(my_vqf, reference_vals, vals, nvals, hits);


	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("fp Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();


  	return diff;
}



template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> bulk_query_timing(templated_vqf<Key, Val, Wrapper> * my_vqf, uint64_t * reference_vals, key_val_pair<Key, Val, Wrapper> * vals, uint64_t nvals, uint64_t * misses){



	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));


	uint64_t num_blocks = my_vqf->get_num_blocks();

	uint64_t num_teams = my_vqf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	
	my_vqf->attach_lossy_buffers(reference_vals, vals, nvals, num_blocks);
	my_vqf->bulk_query(hits, num_teams);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("Bulk Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu\n", misses[0]);

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();

  	return diff;
}


template <typename Key, typename Val = empty, template<typename T> typename Wrapper = empty_wrapper>
__host__ std::chrono::duration<double> fp_timing(templated_vqf<Key, Val, Wrapper> * my_vqf, uint64_t * reference_vals, key_val_pair<Key, Val, Wrapper> * vals, uint64_t nvals, uint64_t * misses){




	bool * hits;

	hipMalloc((void **) & hits, nvals*sizeof(bool));


	uint64_t num_blocks = my_vqf->get_num_blocks();

	uint64_t num_teams = my_vqf->get_num_teams();

	hipDeviceSynchronize();

	auto start = std::chrono::high_resolution_clock::now();


	
	my_vqf->attach_lossy_buffers(reference_vals, vals, nvals, num_blocks);
	my_vqf->bulk_query(hits, num_teams);

	hipDeviceSynchronize();
	//and insert

	auto end = std::chrono::high_resolution_clock::now();



	//check hits

	check_hits<<<(nvals - 1)/ 1024 + 1, 1024>>>(hits, misses, nvals);

	hipDeviceSynchronize();

	hipFree(hits);



	//check hits

  	std::chrono::duration<double> diff = end-start;


  	std::cout << "FP Queried " << nvals << " in " << diff.count() << " seconds\n";

  	printf("FP Sorted Bulk Queries per second: %f\n", nvals/diff.count());

  	printf("Misses %llu, ratio: %f\n", misses[0], 1.0 * (nvals - misses[0])/nvals);  

  	hipDeviceSynchronize();

  	misses[0] = 0;

  	hipDeviceSynchronize();


  	return diff;
}


template <typename T>
__host__ T * generate_data(uint64_t nitems){


	//malloc space

	T * vals = (T *) malloc(nitems * sizeof(T));


	//			   100,000,000
	uint64_t cap = 100000000ULL;

	for (uint64_t to_fill = 0; to_fill < nitems; to_fill+=0){

		uint64_t togen = (nitems - to_fill > cap) ? cap : nitems - to_fill;


		RAND_bytes((unsigned char *) (vals + to_fill), togen * sizeof(T));



		to_fill += togen;

		printf("Generated %llu/%llu\n", to_fill, nitems);

	}

	return vals;
}

template <typename T>
__host__ T * load_main_data(uint64_t nitems){


	char main_location[] = "/global/cscratch1/sd/hunterm/vqf_data/main_data-32-data.txt";

	//char main_location[] = "/pscratch/sd/h/hunterm/vqf_data/main_data-32-data.txt";

	char * vals = (char * ) malloc(nitems * sizeof(T));

	//std::ifstream myfile(main_location);

	//std::string line;


	FILE * pFile;


	pFile = fopen(main_location, "rb");

	if (pFile == NULL) abort();

	size_t result;

	result = fread(vals, 1, nitems*sizeof(T), pFile);

	if (result != nitems*sizeof(T)) abort();



	// //current supported format is no spacing one endl for the file terminator.
	// if (myfile.is_open()){


	// 	getline(myfile, line);

	// 	strncpy(vals, line.c_str(), sizeof(uint64_t)*nitems);

	// 	myfile.close();
		

	// } else {

	// 	abort();
	// }


	return (T *) vals;


}

template <typename T>
__host__ T * load_alt_data(uint64_t nitems){


	char main_location[] = "/global/cscratch1/sd/hunterm/vqf_data/fp_data-32-data.txt";

	//char main_location[] = "/pscratch/sd/h/hunterm/vqf_data/fp_data-32-data.txt";


	char * vals = (char * ) malloc(nitems * sizeof(T));


	//std::ifstream myfile(main_location);

	//std::string line;


	FILE * pFile;


	pFile = fopen(main_location, "rb");

	if (pFile == NULL) abort();

	size_t result;

	result = fread(vals, 1, nitems*sizeof(T), pFile);

	if (result != nitems*sizeof(T)) abort();



	return (T *) vals;


}

int main(int argc, char** argv) {
	

	uint64_t nbits = atoi(argv[1]);

	uint64_t num_batches = atoi(argv[2]);

	double batch_percent = 1.0 / num_batches;


	uint64_t nitems = (1ULL << nbits) * .85;


	//add one? just to guarantee that the clip is correct
	uint64_t items_per_batch = 1.05*nitems * batch_percent;


	printf("Starting test with %d bits, %llu items inserted in %d batches of %d.\n", nbits, nitems, num_batches, items_per_batch);


	using key_type = uint16_t;
	using main_data_type = key_val_pair<key_type>;

	uint64_t * val_references;
	uint64_t * dev_val_references;


	main_data_type * vals;
	main_data_type * dev_vals;


	val_references = load_main_data<uint64_t>(nitems);

	vals = load_main_data<main_data_type>(nitems);


	uint64_t * fp_val_references;

	main_data_type * fp_vals;

	//generate fp data to see comparison with true inserts
	fp_vals = load_alt_data<main_data_type>(nitems);

	fp_val_references = load_alt_data<uint64_t>(nitems);

	// vals = (uint64_t*) malloc(nitems*sizeof(vals[0]));

	// RAND_bytes((unsigned char *)vals, sizeof(*vals) * nitems);


	// other_vals = (uint64_t*) malloc(nitems*sizeof(other_vals[0]));

	// RAND_bytes((unsigned char *)other_vals, sizeof(*other_vals) * nitems);




	hipMalloc((void ** )& dev_vals, items_per_batch*sizeof(main_data_type));

	hipMalloc((void ** )& dev_val_references, items_per_batch*sizeof(uint64_t));

	//hipMemcpy(dev_vals, vals, nitems * sizeof(vals[0]), hipMemcpyHostToDevice);


	//bool * inserts;


	// hipMalloc((void ** )& inserts, items_per_batch*sizeof(bool));

	// hipMemset(inserts, 0, items_per_batch*sizeof(bool));



	// hipMalloc((void ** )& dev_other_vals, nitems*sizeof(other_vals[0]));

	// hipMemcpy(dev_other_vals, other_vals, nitems * sizeof(other_vals[0]), hipMemcpyHostToDevice);


	//allocate misses counter
	uint64_t * misses;
	hipMallocManaged((void **)& misses, sizeof(uint64_t));

	misses[0] = 0;


	//change the way vqf is built to better suit test and use cases? TODO with active reconstruction for exact values / struct support
	
	//quad_hash_table * ht =  build_hash_table(1ULL << nbits);
	templated_vqf<key_type> * vqf = build_vqf<key_type>( (uint64_t)(1ULL << nbits));

	// std::chrono::duration<double>  * insert_diff = std::chrono::nanoseconds::zero();
	// std::chrono::duration<double>  * query_diff = std::chrono::nanoseconds::zero();
	// std::chrono::duration<double>  * fp_diff = std::chrono::nanoseconds::zero();

	std::chrono::duration<double>  * insert_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
	std::chrono::duration<double>  * query_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));
	std::chrono::duration<double>  * fp_diff = (std::chrono::duration<double>  *) malloc(num_batches*sizeof(std::chrono::duration<double>));

	uint64_t * batch_amount = (uint64_t *) malloc(num_batches*sizeof(uint64_t));

	printf("Setup done\n");

	//wipe_vals<<<nitems/32+1, 32>>>(dev_vals, nitems);


	hipDeviceSynchronize();

	

	for (int batch = 0; batch< num_batches; batch++){

		//calculate size of segment

		printf("Batch %d:\n", batch);

		//runs from batch/num_batches*nitems to batch
		uint64_t start = batch*nitems/num_batches;
		uint64_t end = (batch+1)*nitems/num_batches;
		if (end > nitems) end = nitems;

		uint64_t items_to_insert = end-start;

		batch_amount[batch] = items_to_insert;


		assert(items_to_insert < items_per_batch);

		//prep dev_vals for this round

		hipMemcpy(dev_val_references, val_references + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);

		hipMemcpy(dev_vals, vals + start, items_to_insert*sizeof(main_data_type), hipMemcpyHostToDevice);

		hipDeviceSynchronize();

		//launch inserts
		//diff += split_insert_timing<key_type>(vqf, dev_val_references, dev_vals, items_to_insert, misses);
		insert_diff[batch] = split_insert_timing<key_type>(vqf, dev_val_references, dev_vals, items_to_insert, misses);

		//insert_diff[batch] = single_insert_timing<key_type>(vqf, dev_val_references, dev_vals, items_to_insert, misses);


		hipDeviceSynchronize();

		hipMemcpy(dev_val_references, val_references + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);

		hipMemcpy(dev_vals, vals + start, items_to_insert*sizeof(main_data_type), hipMemcpyHostToDevice);

		hipDeviceSynchronize();


		//launch queries
		//query_diff[batch] = single_bulk_query_timing<key_type>(vqf, dev_val_references, dev_vals, items_to_insert, misses);
		query_diff[batch] = bulk_query_timing<key_type>(vqf, dev_val_references, dev_vals, items_to_insert, misses);


		hipDeviceSynchronize();

		hipMemcpy(dev_val_references, fp_val_references + start, items_to_insert*sizeof(uint64_t), hipMemcpyHostToDevice);

		hipMemcpy(dev_vals, fp_vals + start, items_to_insert*sizeof(main_data_type), hipMemcpyHostToDevice);

		hipDeviceSynchronize();


		//false queries
		fp_diff[batch] =  fp_timing<key_type>(vqf, dev_val_references, dev_vals, items_to_insert, misses);
		//fp_diff[batch] =  single_fp_timing<key_type>(vqf, dev_val_references, dev_vals, items_to_insert, misses);

		hipDeviceSynchronize();


		//keep some organized spacing
		printf("\n\n");

		fflush(stdout);

		hipDeviceSynchronize();



	}


	std::chrono::duration<double> summed_insert_diff = std::chrono::nanoseconds::zero();

	for (int i =0; i < num_batches;i++){
		summed_insert_diff += insert_diff[i];
	}

	std::chrono::duration<double> summed_query_diff = std::chrono::nanoseconds::zero();

	for (int i =0; i < num_batches;i++){
		summed_query_diff += query_diff[i];
	}

	std::chrono::duration<double> summed_fp_diff = std::chrono::nanoseconds::zero();

	for (int i =0; i < num_batches;i++){
		summed_fp_diff += fp_diff[i];
	}

	printf("Tests Finished.\n");

	std::cout << "Queried " << nitems << " in " << summed_insert_diff.count() << " seconds\n";

	printf("Final speed: %f\n", nitems/summed_insert_diff.count());


	if (argc == 4){

		printf("Dumping into file\n");

		const char * dir = "batched_results/";

		char filename_insert[256];
		char filename_lookup[256];
		char filename_false_lookup[256];
		char filename_aggregate[256];

		const char * insert_op = "_insert_";

		snprintf(filename_insert, strlen(dir) + strlen(argv[3]) + strlen(insert_op) + strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], insert_op, argv[1], argv[2]);

		const char * lookup_op = "_lookup_";

		snprintf(filename_lookup, strlen(dir) + strlen(argv[3]) + strlen(lookup_op) + strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], lookup_op, argv[1], argv[2]);

		const char * fp_ops = "_fp_";

		snprintf(filename_false_lookup, strlen(dir) + strlen(argv[3]) + strlen(fp_ops) + strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], fp_ops, argv[1], argv[2]);

		const char * agg_ops = "_aggregate_";

		snprintf(filename_aggregate, strlen(dir) + strlen(argv[3]) + strlen(agg_ops)+ strlen(argv[1]) + strlen(argv[2]) + 2, "%s%s%s%s_%s", dir, argv[3], agg_ops, argv[1], argv[2]);


		FILE *fp_insert = fopen(filename_insert, "w");
		FILE *fp_lookup = fopen(filename_lookup, "w");
		FILE *fp_false_lookup = fopen(filename_false_lookup, "w");
		FILE *fp_agg = fopen(filename_aggregate, "w");

		if (fp_insert == NULL) {
			printf("Can't open the data file %s\n", filename_insert);
			exit(1);
		}

		if (fp_lookup == NULL ) {
		    printf("Can't open the data file %s\n", filename_lookup);
			exit(1);
		}

		if (fp_false_lookup == NULL) {
			printf("Can't open the data file %s\n", filename_false_lookup);
			exit(1);
		}

		if (fp_agg == NULL) {
			printf("Can't open the data file %s\n", filename_aggregate);
			exit(1);
		}


		printf("Writing results to file: %s\n",  filename_insert);

		fprintf(fp_insert, "x_0 y_0\n");
		for (int i = 0; i < num_batches; i++){
			fprintf(fp_insert, "%d", i*100/num_batches);

			fprintf(fp_insert, " %f\n", batch_amount[i]/insert_diff[i].count());
		}
		printf("Insert performance written!\n");

		fclose(fp_insert);


		printf("Writing results to file: %s\n",  filename_lookup);

		fprintf(fp_lookup, "x_0 y_0\n");
		for (int i = 0; i < num_batches; i++){
			fprintf(fp_lookup, "%d", i*100/num_batches);

			fprintf(fp_lookup, " %f\n", batch_amount[i]/query_diff[i].count());
		}
		printf("lookup performance written!\n");

		fclose(fp_lookup);



		printf("Writing results to file: %s\n",  filename_false_lookup);

		fprintf(fp_false_lookup, "x_0 y_0\n");
		for (int i = 0; i < num_batches; i++){
			fprintf(fp_false_lookup, "%d", i*100/num_batches);

			fprintf(fp_false_lookup, " %f\n", batch_amount[i]/fp_diff[i].count());
		}
		printf("false_lookup performance written!\n");

		fclose(fp_false_lookup);


		printf("Writing results to file: %s\n",  filename_aggregate);

		//fprintf(fp_agg, "x_0 y_0\n");

		fprintf(fp_agg, "Aggregate inserts: %f\n", nitems/summed_insert_diff.count());
		fprintf(fp_agg, "Aggregate Queries: %f\n", nitems/summed_query_diff.count());
		fprintf(fp_agg, "Aggregate fp: %f\n", nitems/summed_fp_diff.count());



		printf("false_lookup performance written!\n");

		fclose(fp_false_lookup);



	}


	free(vals);

	free(fp_vals);

	hipFree(dev_vals);

	hipFree(misses);

	free_vqf(vqf);

	

	return 0;

}
