#include "hip/hip_runtime.h"

#ifndef _ATOMIC_BLOCK_CU
#define _ATOMIC_BLOCK_CU


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "include/atomic_block.cuh"
#include "include/warp_utils.cuh"
#include "include/metadata.cuh"

//extra stuff
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>

//VQF Block
// Functions Required:

// Lock();
// get_fill();
// Unlock_other();
// Insert();
// Unlock();

//I'm putting the bit manipulation here atm



//set the original 1 bits of the block
//this is done on a per thread level
__device__ void atomic_block::setup(){

	md = 0;

	#if DEBUG_ASSERTS

	//verify that the blocks are aligned according to their idea of a cache line
	assert(sizeof(atomic_block) % BYTES_PER_CACHE_LINE == 0);

	#endif
}






//return the number of filled slots
//much easier to get the number of unfilled slots
//and do capacity - unfilled
//do a trailing zeros count on the rightmost md counter
__device__ int atomic_block::get_fill(){

	return md;

}

__device__ int atomic_block::get_fill_atomic(){

	return atomicCAS((unsigned int * ) &md, (unsigned int) 0, (unsigned int) 0);

}

__device__ int atomic_block::max_capacity(){

	return SLOTS_PER_BLOCK;
}



//atomicAdd to grab the next available slot
__device__ void atomic_block::insert(int warpID, uint64_t item){

	
	if (warpID == 0){
		insert_one_thread(item);
	}
	return;

}

__device__ bool atomic_block::query(int warpID, uint64_t item){

	#if TAG_BITS == 8
		uint8_t tag = item & 0xFF;

	#elif TAG_BITS == 16
		uint16_t tag = item & 0xFFFF;
	#endif

	if (tag == TOMBSTONE_VAL){
		tag += 1;
	}

	int fill = get_fill();

	int ballot = 0;


	for (int i = warpID; i < fill; i+=32){

		if (tags[i] == tag) ballot = 1;
	}

	// if (warpID < fill){


	// 	if (tags[warpID] == tag) ballot = 1;
	// }


	unsigned int ballot_result = __ballot_sync(0xffffffff, ballot);

	int thread_to_query = __ffs(ballot_result) -1;


	if (thread_to_query == -1) return false;

	return true;


}


//attempt to remove an item with tag item, 
//returns false if no item to remove
__device__ bool atomic_block::remove(int warpID, uint64_t item){



	//TODO: this


	//return false;


	#if TAG_BITS == 8
		uint8_t tag = item & 0xFF;
	#elif TAG_BITS == 16
		uint16_t tag = item & 0xFFFF;
	#endif


	int fill = get_fill();

	int ballot = 0;


	//the old one has a problem where multiple items could be deleted

	for (int i =0; i < fill; i+=32){

		int my_slot = i + warpID;


		if (my_slot < fill){
			if (tags[my_slot] == tag) ballot = 1;
		}

		


		unsigned int ballot_result = __ballot_sync(0xffffffff, ballot);

		int thread_to_query = __ffs(ballot_result) -1;

		if (thread_to_query != -1){


			//end
			if (thread_to_query == warpID){

				//change this later
				tags[my_slot] = TOMBSTONE_VAL;
			}

		}




	}


}

__device__ bool atomic_block::purge_tombstone(int warpID){

	int fill = get_fill();


	//keep track of tombstones and non-tombstones, non-tombstones go to the front and tombstones are subtracted from the main list. 
	int non_tombstones =0;

	int tombstones = 0;

	for (int i = warpID; i < fill; i+= 32){

		if (tags[i] == TOMBSTONE_VAL){

			tombstones += 1;
		} else {
			non_tombstones += 1;
		}

	}

	int start= non_tombstones;

	//sync and ballot here
		for (int i=1; i<=16; i*=2) {
	    // We do the __shfl_sync unconditionally so that we
	    // can read even from threads which won't do a
	    // sum, and then conditionally assign the result.
	    int n = __shfl_up_sync(0xffffffff, start, i, 32);
	    if ((warpID) >= i)
	        start += n;
	}

	//read the thread "ahead" of me
	int prev = __shfl_up_sync(0xffffffff, start, 1, 32);

	if (warpID == 0){
		prev = 0;
	} 

	//use prev as start

	#if DEBUG_ASSERTS

	assert(non_tombstones <= 8);

	#endif

	#if TAG_BITS == 16 

	uint16_t temp_bits[8];

	int temp_start = 0;

	#else 

	uint8_t temp_bits[8];

	int temp_start = 0;

	#endif

	for (int i = warpID; i < fill; i++){

		if (tags[i] != TOMBSTONE_VAL){

			temp_bits[temp_start] = tags[i];
			temp_start+=1;
		}

	}

	//regroup for write
	__syncwarp();

	atomicSub((unsigned int *) & md, (unsigned int) tombstones);

	for (int i=0; i < temp_start; i++){

		tags[prev + i] = temp_bits[i];
	}



}


__device__ bool atomic_block::insert_one_thread(uint64_t item){

		#if TAG_BITS == 8
			uint8_t tag = item & 0xFF;
		#elif TAG_BITS == 16
			uint16_t tag = item & 0xFFFF;
		#endif

		if (tag == TOMBSTONE_VAL){
			tag += 1;
		}

		int fill = atomicAdd((unsigned int *) &md, (unsigned int) 1);

		if (fill < SLOTS_PER_BLOCK){
			tags[fill] = tag;
		} else {
			//undo addition so that removes function as expected
			atomicSub((unsigned int *) & md, (unsigned int) 1);
			return false;
		}

		

	__threadfence();


	return true;
}


__device__ void atomic_block::bulk_insert(int warpID, uint64_t * items, uint64_t nitems){

	

	// - slot necessary - the buckets are logical constructs
	//and don't correspond to true indices.


	//for the 16 bit 64 byte case maybe write a preprocessor directive to not do the loup

	int fill = get_fill();

	for (int i = warpID; i < nitems; i+=32){

		uint64_t item = items[i];


		#if TAG_BITS == 8
			uint8_t tag = item & 0xFF;
		#elif TAG_BITS == 16
			uint16_t tag = item & 0xFFFF;
		#endif

		if (tag == TOMBSTONE_VAL){
			tag += 1;
		}


		tags[i + fill] = tag;
		

	}
	


	if (warpID == 0) atomicAdd((unsigned int *) & md, nitems);


	__syncwarp();

	return;

}



//TODO: Patch this
//BUlk Query can only find items that are < 32
__device__ int atomic_block::bulk_query(int warpID, uint64_t * items, uint64_t nitems){

	#if DEBUG_ASSERTS

	assert(nitems < 32);

	#endif

	uint64_t item =0;

	if (warpID < nitems) item = items[warpID];

	#if TAG_BITS == 8
		uint8_t tag = item & 0xFF;
	#elif TAG_BITS == 16
		uint16_t tag = item & 0xFFFF;
	#endif

	if (tag == TOMBSTONE_VAL){
		tag += 1;
	}

	int ballot = 0;

	if (warpID < nitems){


		for (int i = 0; i < nitems; i++){
			if (tags[i] == tag) {
				ballot = 1;
				break;
			}
		}
	}
	__syncwarp();

	unsigned int ballot_result = __ballot_sync(0xffffffff, ballot);

	return __popc(ballot_result);



}

__device__ bool atomic_block::assert_consistency(){


	if (md <= SLOTS_PER_BLOCK) return true;

	return false;

}





#endif //atomic_block_CU