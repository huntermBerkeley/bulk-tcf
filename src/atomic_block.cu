#include "hip/hip_runtime.h"

#ifndef _ATOMIC_BLOCK_CU
#define _ATOMIC_BLOCK_CU


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "include/atomic_block.cuh"
#include "include/warp_utils.cuh"
#include "include/metadata.cuh"
#include "include/sorting_helper.cuh"

//extra stuff
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <time.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <random>
#include <assert.h>
#include <chrono>
#include <iostream>

#include <fstream>
#include <string>
#include <algorithm>
#include <bitset>

//VQF Block
// Functions Required:

// Lock();
// get_fill();
// Unlock_other();
// Insert();
// Unlock();

//I'm putting the bit manipulation here atm



//set the original 1 bits of the block
//this is done on a per thread level
__device__ void atomic_block::setup(){

	md = 0;

	#if DEBUG_ASSERTS

	//verify that the blocks are aligned according to their idea of a cache line
	assert(sizeof(atomic_block) % BYTES_PER_CACHE_LINE == 0);

	#endif
}






//return the number of filled slots
//much easier to get the number of unfilled slots
//and do capacity - unfilled
//do a trailing zeros count on the rightmost md counter
__device__ int atomic_block::get_fill(){

	return md;

}

__device__ int atomic_block::get_fill_atomic(){

	return atomicCAS((unsigned int * ) &md, (unsigned int) 0, (unsigned int) 0);

}

__device__ int atomic_block::max_capacity(){

	return SLOTS_PER_BLOCK;
}



//atomicAdd to grab the next available slot
__device__ void atomic_block::insert(int warpID, uint64_t item){

	
	if (warpID == 0){
		insert_one_thread(item);
	}
	return;

}

__device__ bool atomic_block::query(int warpID, uint64_t item){


	#if TAG_BITS == 8
		uint8_t tag = item & 0xFF;

	#elif TAG_BITS == 16
		uint16_t tag = item & 0xFFFF;
	#endif

	if (tag == TOMBSTONE_VAL){
		tag += 1;
	}

	int fill = get_fill();

//	int fill_cutoff = ((fill -1)/32 + 1) * 32;

	int ballot = 0;


	for (int i = warpID; i < fill; i+=32){

		if (tags[i] == tag) ballot = 1;
	}

	// if (warpID < fill){


	// 	if (tags[warpID] == tag) ballot = 1;
	// }


	unsigned int ballot_result = __ballot_sync(0xffffffff, ballot);

	int thread_to_query = __ffs(ballot_result) -1;


	if (thread_to_query == -1) return false;

	return true;


	// for (int i = warpID; i < fill_cutoff; i+=32){

	// 	if (i < fill && tags[i] == tag) ballot = 1;

	// 	unsigned int ballot_result = __ballot_sync(0xffffffff, ballot);

	// 	int thread_to_query = __ffs(ballot_result) -1;


	// 	if (thread_to_query != -1) return true;
	// }


	// return false;


}


//attempt to remove an item with tag item, 
//returns false if no item to remove
__device__ bool atomic_block::remove(int warpID, uint64_t item){



	//TODO: this


	//return false;


	#if TAG_BITS == 8
		uint8_t tag = item & 0xFF;
	#elif TAG_BITS == 16
		uint16_t tag = item & 0xFFFF;
	#endif


	int fill = get_fill();

	int ballot = 0;


	//the old one has a problem where multiple items could be deleted

	for (int i =0; i < fill; i+=32){

		int my_slot = i + warpID;


		if (my_slot < fill){
			if (tags[my_slot] == tag) ballot = 1;
		}

		


		unsigned int ballot_result = __ballot_sync(0xffffffff, ballot);

		int thread_to_query = __ffs(ballot_result) -1;

		if (thread_to_query != -1){


			//end
			if (thread_to_query == warpID){

				//change this later
				tags[my_slot] = TOMBSTONE_VAL;
			}

		}




	}


}

__device__ bool atomic_block::purge_tombstone(int warpID){

	int fill = get_fill();


	//keep track of tombstones and non-tombstones, non-tombstones go to the front and tombstones are subtracted from the main list. 
	int non_tombstones =0;

	int tombstones = 0;

	for (int i = warpID; i < fill; i+= 32){

		if (tags[i] == TOMBSTONE_VAL){

			tombstones += 1;
		} else {
			non_tombstones += 1;
		}

	}

	int start= non_tombstones;

	//sync and ballot here
		for (int i=1; i<=16; i*=2) {
	    // We do the __shfl_sync unconditionally so that we
	    // can read even from threads which won't do a
	    // sum, and then conditionally assign the result.
	    int n = __shfl_up_sync(0xffffffff, start, i, 32);
	    if ((warpID) >= i)
	        start += n;
	}

	//read the thread "ahead" of me
	int prev = __shfl_up_sync(0xffffffff, start, 1, 32);

	if (warpID == 0){
		prev = 0;
	} 

	//use prev as start

	#if DEBUG_ASSERTS

	assert(non_tombstones <= 8);

	#endif

	#if TAG_BITS == 16 

	uint16_t temp_bits[8];

	int temp_start = 0;

	#else 

	uint8_t temp_bits[8];

	int temp_start = 0;

	#endif

	for (int i = warpID; i < fill; i++){

		if (tags[i] != TOMBSTONE_VAL){

			temp_bits[temp_start] = tags[i];
			temp_start+=1;
		}

	}

	//regroup for write
	__syncwarp();

	atomicSub((unsigned int *) & md, (unsigned int) tombstones);

	for (int i=0; i < temp_start; i++){

		tags[prev + i] = temp_bits[i];
	}



}


__device__ bool atomic_block::insert_one_thread(uint64_t item){

		#if TAG_BITS == 8
			uint8_t tag = item & 0xFF;
		#elif TAG_BITS == 16
			uint16_t tag = item & 0xFFFF;
		#endif

		if (tag == TOMBSTONE_VAL){
			tag += 1;
		}

		int fill = atomicAdd((unsigned int *) &md, (unsigned int) 1);

		if (fill < SLOTS_PER_BLOCK){
			tags[fill] = tag;
		} else {
			//undo addition so that removes function as expected
			atomicSub((unsigned int *) & md, (unsigned int) 1);
			return false;
		}

		

	__threadfence();


	return true;
}


__device__ void atomic_block::bulk_insert(int warpID, uint64_t * items, uint64_t nitems){

	

	// - slot necessary - the buckets are logical constructs
	//and don't correspond to true indices.


	//for the 16 bit 64 byte case maybe write a preprocessor directive to not do the loup

	int fill = get_fill();

	for (int i = warpID; i < nitems; i+=32){

		uint64_t item = items[i];


		#if TAG_BITS == 8
			uint8_t tag = item & 0xFF;
		#elif TAG_BITS == 16
			uint16_t tag = item & 0xFFFF;
		#endif

		if (tag == TOMBSTONE_VAL){
			tag += 1;
		}


		tags[i + fill] = tag;
		

	}
	


	if (warpID == 0) atomicAdd((unsigned int *) & md, nitems);


	__syncwarp();

	return;

}


#if TAG_BITS == 8

	__device__ void atomic_block::sorted_bulk_insert(uint8_t * temp_tags, uint64_t * items, uint64_t nitems, int teamID, int warpID)


#elif TAG_BITS == 16

	__device__ void atomic_block::sorted_bulk_insert(uint16_t * temp_tags, uint64_t * items, uint64_t nitems, int teamID, int warpID)


#endif

	{	

	
	//for the 16 bit 64 byte case maybe write a preprocessor directive to not do the loop



	int fill = get_fill();


	//without debug on you can mess this up, safety checks are handled at that level by higher up
	//processes
	#if DEBUG_ASSERTS


	

	#if TAG_BITS == 8

	assert(byte_assert_sorted(items, nitems));

	assert(short_byte_assert_sorted(tags, fill));

	#elif TAG_BITS == 16

	if (!two_byte_assert_sorted(items, nitems)){

		assert(two_byte_assert_sorted(items, nitems));

	}



	assert(sixteen_byte_assert_sorted(tags, fill));

	#endif



	#endif



	//now that bounds are checked, setup for main insert


	#if TAG_BITS == 8

	merge_dual_arrays_8_bit_64_bit(temp_tags, &tags[0], items, fill, nitems, teamID, warpID);


	#elif TAG_BITS == 16

	merge_dual_arrays_16_bit_64_bit(temp_tags, &tags[0], items, fill, nitems, teamID, warpID);

	#endif


	


	if (warpID == 0) atomicAdd((unsigned int *) & md, nitems);



	__syncwarp();


	#if DEBUG_ASSERTS



	#if TAG_BITS == 8


	if (!short_byte_assert_sorted(tags, fill+nitems)){

		assert(short_byte_assert_sorted(tags, fill+nitems));

	}


	#elif TAG_BITS == 16

	if (!sixteen_byte_assert_sorted(tags, fill+nitems)){

		assert(sixteen_byte_assert_sorted(tags, fill+nitems));

	}

	#endif

	#endif



	return;

}


//NEW STUFF - dynamic write back to main memory

#if TAG_BITS == 8

	__device__ void atomic_block::dump_all_buffers_sorted(uint64_t * global_buffer, int buffer_count, uint8_t * original_items, int nitems, uint8_t * remaining_items, int n_remaining, int teamID, int warpID);


#elif TAG_BITS == 16

	__device__ void atomic_block::dump_all_buffers_sorted(uint64_t * global_buffer, int buffer_count, uint16_t * original_items, int nitems, uint16_t * remaining_items, int n_remaining, int teamID, int warpID)

#endif

	{	

	
	//for the 16 bit 64 byte case maybe write a preprocessor directive to not do the loop


	//who cares about fill we don't need it! we just need to update with the appropriate length at the end
	//int fill = get_fill();


	//without debug on you can mess this up, safety checks are handled at that level by higher up
	//processes
	#if DEBUG_ASSERTS


	assert(buffer_count+n_remaining+nitems <= SLOTS_PER_BLOCK);
	

	#if TAG_BITS == 8

	assert(byte_assert_sorted(global_buffer, buffer_count));

	assert(short_byte_assert_sorted(original_items, nitems));

	assert(short_byte_assert_sorted(remaining_list, n_remaining));

	#elif TAG_BITS == 16

	if (!two_byte_assert_sorted(global_buffer, buffer_count)){

		assert(two_byte_assert_sorted(global_buffer, buffer_count));

	}



	//assert(sixteen_byte_assert_sorted(original_list, n_list));

	assert(sixteen_byte_assert_sorted(original_items, nitems));

	assert(sixteen_byte_assert_sorted(remaining_items, n_remaining));

	#endif



	#endif



	//now that bounds are checked, setup for main insert


	#if TAG_BITS == 8

	abort();

	#elif TAG_BITS == 16

	merge_3_into_tags_16_bit(&tags[0], global_buffer, buffer_count, original_items, nitems, remaining_items, n_remaining, teamID, warpID);

	#endif


	

	//entirely replace
	//double check on this as well
	if (warpID == 0) atomicExch((unsigned int *) & md, nitems+buffer_count+n_remaining);



	__syncwarp();


	#if DEBUG_ASSERTS


	int fill = get_fill();

	assert(fill <= SLOTS_PER_BLOCK);

	#if TAG_BITS == 8


	if (!short_byte_assert_sorted(tags, fill)){

		assert(short_byte_assert_sorted(tags, fill));

	}


	#elif TAG_BITS == 16

	if (!sixteen_byte_assert_sorted(tags, fill)){

		assert(sixteen_byte_assert_sorted(tags, fill));

	}

	#endif

	#endif



	return;

 }


//a variant of the insert scheme that treats temp_tags as a local array, because it is
//this is a workaround to redefining the shared memory structure of the entire project
// while still maintaining minimal memory use

#if TAG_BITS == 8

__device__ void atomic_block::sorted_bulk_finish(uint8_t * temp_tags, uint8_t * items, uint64_t nitems, int teamID, int warpID)


#elif TAG_BITS == 16

__device__ void atomic_block::sorted_bulk_finish(uint16_t * temp_tags, uint16_t * items, uint64_t nitems, int teamID, int warpID)


#endif

	{


	int fill = get_fill();

	#if DEBUG_ASSERTS


	#if TAG_BITS == 8

	assert(short_byte_assert_sorted(items, nitems));

	assert(short_byte_assert_sorted(tags, fill));


	#elif TAG_BITS == 16

	assert(sixteen_byte_assert_sorted(items, nitems));

	assert(sixteen_byte_assert_sorted(tags, fill));

	#endif



	if (nitems + fill >= SLOTS_PER_BLOCK){

		assert(nitems + fill <= SLOTS_PER_BLOCK);
	}
	


	#endif


	//now that bounds are checked, setup for main insert

	//TODO fix merge_dual_arrays

	#if TAG_BITS == 8
	merge_dual_arrays(temp_tags, &tags[0], items, fill, nitems, teamID, warpID);


	#elif TAG_BITS == 16

	merge_dual_arrays_sixteen(temp_tags, &tags[0], items, fill, nitems, teamID, warpID);

	#endif


	


	if (warpID == 0) atomicAdd((unsigned int *) & md, nitems);



	__syncwarp();


	#if DEBUG_ASSERTS

	#if TAG_BITS == 8

	if (!short_byte_assert_sorted(tags, fill+nitems)){

		assert(short_byte_assert_sorted(tags, fill+nitems));

	}

	#elif TAG_BITS == 16

	if (!sixteen_byte_assert_sorted(tags, fill+nitems)){

		assert(sixteen_byte_assert_sorted(tags, fill+nitems));

	}

	#endif

	#endif

	return;







}



//TODO: Patch this
//BUlk Query can only find items that are < 32
__device__ int atomic_block::bulk_query(int warpID, uint64_t * items, uint64_t nitems){

	#if DEBUG_ASSERTS

	assert(nitems < 32);

	#endif

	uint64_t item =0;

	if (warpID < nitems) item = items[warpID];

	#if TAG_BITS == 8
		uint8_t tag = item & 0xFF;
	#elif TAG_BITS == 16
		uint16_t tag = item & 0xFFFF;
	#endif

	if (tag == TOMBSTONE_VAL){
		tag += 1;
	}

	int ballot = 0;

	if (warpID < nitems){


		for (int i = 0; i < nitems; i++){
			if (tags[i] == tag) {
				ballot = 1;
				break;
			}
		}
	}
	__syncwarp();

	unsigned int ballot_result = __ballot_sync(0xffffffff, ballot);

	return __popc(ballot_result);



}

__device__ bool atomic_block::assert_consistency(){


	if (md <= SLOTS_PER_BLOCK) return true;

	return false;

}





//replace this with a recursive bitonic sort
__device__ bool atomic_block::sort_block(int teamID, int warpID){


	// int fill = get_fill();

	// shortByteBitonicSort(tags, 0, fill, true, warpID);

	// __syncwarp();

	int fill = get_fill();

	//short_warp_sort(tags, fill, teamID, warpID);

	//bubble_sort(tags, fill, warpID);


	// while (true){


	// 	bool sorted = false;

	// 	//even transpositions
	// 	for (int i = warpID*2+1; i < fill; i+=64){

	// 		//swap warpID*2, warpID*2+1

	// 		if ((tags[i-1] & 0xFF) > (tags[i] & 0xFF)){

	// 			#if TAG_BITS == 8

	// 			uint8_t temp_tag;

	// 			#else

	// 			uint16_t temp_tag;

	// 			#endif

	// 			temp_tag = tags[i-1];

	// 			tags[i-1] = tags[i];

	// 			tags[i] = temp_tag;

	// 			sorted = true;

	// 		}



	// 	}


	// 	//odd transpositions
	// 	for (int i = warpID*2+2; i < fill; i+=64){

	// 		//swap warpID*2, warpID*2+1

	// 		if ((tags[i-1] & 0xFF) > (tags[i] & 0xFF)){

	// 			#if TAG_BITS == 8

	// 			uint8_t temp_tag;

	// 			#else

	// 			uint16_t temp_tag;

	// 			#endif

	// 			temp_tag = tags[i-1];

	// 			tags[i-1] = tags[i];

	// 			tags[i] = temp_tag;

	// 			sorted = true;

	// 		}



	// 	}

	// 	if (__ffs(__ballot_sync(0xffffffff, sorted)) == 0) return;


	// }


}


//this is a check, no fancy schmancyness
__device__ bool atomic_block::assert_sorted(int warpID){


	int fill = get_fill();


	#if TAG_BITS == 8

		return short_byte_assert_sorted(tags, fill);

	#elif TAG_BITS == 16

		return sixteen_byte_assert_sorted(tags, fill);

	#endif



}



//inner sorted join
//assume both are prepped and sorted
//ill fix the comparison shit later
__device__ bool atomic_block::sorted_bulk_query(int warpID, uint64_t * items, bool * found, uint64_t nitems){


	//byteBitonicSort(items, 0, nitems, true, warpID);


	//big_bubble_sort(items, nitems, warpID);


	#if DEBUG_ASSERTS

	#if TAG_BITS == 8

	assert(byte_assert_sorted(items, nitems));

	#elif TAG_BITS == 16

	assert(two_byte_assert_sorted(items, nitems));

	#endif

	#endif

	//bitonicSort(uint64_t * items, int low, int count, bool dir, int warpID){

	int fill = get_fill();

	//bubble_sort(tags, fill, warpID);

	__syncwarp();

	#if DEBUG_ASSERTS

	#if TAG_BITS == 8 

	assert(short_byte_assert_sorted(tags, fill));

	#elif TAG_BITS == 16

	assert(sixteen_byte_assert_sorted(tags, fill));

	#endif

	#endif

	if (fill == 0 || nitems == 0) return;


	int left = 0;
	int right = 0;

	while (true){

		#if TAG_BITS == 8

		uint8_t comp = items[left] & 0xFF;

		#elif TAG_BITS == 16

		uint16_t comp = items[left] & 0xFFFF;

		#endif

		if (comp == tags[right]){

			found[left] = true;
			left++;

			if (left >= nitems) return;


		} else if (comp < tags[right]){

			//left is a miss
			found[left] = false;
			left++;

			if (left >= nitems) return;

		} //else if (items[left] > tags[right])
		else {

			right++;

			if (right >= fill){

				//purge remaining 
				for (int i = left; i < nitems; i++){

					found[i] = false;

				}

				return;

			}

		

		}




	}



} 



//first attempt - query 32 values at a time from the main list
__device__ bool atomic_block::sorted_bulk_query_cooperative(int warpID, uint64_t * items, bool * found, uint64_t nitems){


	//byteBitonicSort(items, 0, nitems, true, warpID);


	//big_bubble_sort(items, nitems, warpID);


	#if DEBUG_ASSERTS

	#if TAG_BITS == 8

	assert(byte_assert_sorted(items, nitems));

	#elif TAG_BITS == 16

	assert(two_byte_assert_sorted(items, nitems));

	#endif

	#endif

	//bitonicSort(uint64_t * items, int low, int count, bool dir, int warpID){

	int fill = get_fill();

	//bubble_sort(tags, fill, warpID);

	__syncwarp();

	#if DEBUG_ASSERTS

	#if TAG_BITS == 8 

	assert(short_byte_assert_sorted(tags, fill));

	#elif TAG_BITS == 16

	assert(sixteen_byte_assert_sorted(tags, fill));

	#endif

	#endif

	if (fill == 0 || nitems == 0) return;


	int left = 0;
	int right = 0;

	while (true){

		#if TAG_BITS == 8

		uint8_t comp = items[left] & 0xFF;

		#elif TAG_BITS == 16

		uint16_t comp = items[left] & 0xFFFF;

		#endif


		//new workflow, everyone checks for an exact match
		//then everyone queries greater than
		//if exact we continue
		//else if greater than we are done

		bool avoiding_segfault = right+warpID < fill;

		bool ballot = (avoiding_segfault && comp == tags[right+warpID]);

		int thread_found = __ffs(__ballot_sync(0xffffffff, ballot)) -1;

		if (thread_found != -1){


			found[left] = true;
			//increment left by one
			left++;

			//since all items < thread_found are < tags[right+warpID], skip items
			right+=thread_found;

			if (left >= nitems) return;


		} else {

			//else purge all items smaller

			bool ballot = (avoiding_segfault && comp > tags[right+warpID]);

			int first_larger = __ffs(__ballot_sync(0xffffffff, ballot));

			//0 2 2 3
			// 1

			//t f f f

			//skip threadID+1

			//if we didn't find you and someone here is smaller, you don't exist
			if (first_larger > 0){
				found[left] = false;
				left++;

				if (left >= nitems) return;

			}

			right+=first_larger;

			if (right >= fill){

				for (int i = left+warpID; i < nitems; i+=32){
					found[i] = false;
				}

				return;
			}


		}






	}



} 


__device__ int atomic_block::sorted_bulk_query_num_found(int warpID, uint64_t * items, uint64_t nitems){


	//byteBitonicSort(items, 0, nitems, true, warpID);

	int num_found = 0;


	//big_bubble_sort(items, nitems, warpID);


	#if DEBUG_ASSERTS

	#if TAG_BITS == 8

	assert(byte_assert_sorted(items, nitems));

	#elif TAG_BITS == 16

	assert(two_byte_assert_sorted(items, nitems));

	#endif

	#endif

	//bitonicSort(uint64_t * items, int low, int count, bool dir, int warpID){

	int fill = get_fill();

	//bubble_sort(tags, fill, warpID);

	__syncwarp();

	#if DEBUG_ASSERTS

	#if TAG_BITS == 8 

	assert(short_byte_assert_sorted(tags, fill));

	#elif TAG_BITS == 16

	assert(sixteen_byte_assert_sorted(tags, fill));

	#endif

	#endif

	if (fill == 0 || nitems == 0) return 0;


	int left = 0;
	int right = 0;

	while (true){

		#if TAG_BITS == 8

		uint8_t comp = items[left] & 0xFF;

		#elif TAG_BITS == 16

		uint16_t comp = items[left] & 0xFFFF;

		#endif

		if (comp == tags[right]){

			num_found++;
			left++;

			if (left >= nitems) return num_found;


		} else if (comp < tags[right]){

			//left is a miss
			num_found++;
			left++;

			if (left >= nitems) return num_found;

		} //else if (items[left] > tags[right])
		else {

			right++;

			if (right >= fill){


				return num_found;

			}

		

		}




	}

	return num_found;



} 



__device__ int atomic_block::sorted_bulk_query_num_found_short(int warpID, uint16_t * items, uint64_t nitems){


	//byteBitonicSort(items, 0, nitems, true, warpID);

	int num_found = 0;


	//big_bubble_sort(items, nitems, warpID);


	#if DEBUG_ASSERTS

	#if TAG_BITS == 8

	assert(short_byte_assert_sorted(items, nitems));

	#elif TAG_BITS == 16

	assert(sixteen_byte_assert_sorted(items, nitems));

	#endif

	#endif

	//bitonicSort(uint64_t * items, int low, int count, bool dir, int warpID){

	int fill = get_fill();

	//bubble_sort(tags, fill, warpID);

	__syncwarp();

	#if DEBUG_ASSERTS

	#if TAG_BITS == 8 

	assert(short_byte_assert_sorted(tags, fill));

	#elif TAG_BITS == 16

	assert(sixteen_byte_assert_sorted(tags, fill));

	#endif

	#endif

	if (fill == 0 || nitems == 0) return 0;


	int left = 0;
	int right = 0;

	while (true){

		#if TAG_BITS == 8

		uint8_t comp = items[left];

		#elif TAG_BITS == 16

		uint16_t comp = items[left];

		#endif

		if (comp == tags[right]){

			num_found++;
			left++;

			if (left >= nitems) return num_found;


		} else if (comp < tags[right]){

			//left is a miss
			num_found++;
			left++;

			if (left >= nitems) return num_found;

		} //else if (items[left] > tags[right])
		else {

			right++;

			if (right >= fill){


				return num_found;

			}

		

		}




	}

	return num_found;



} 



__device__ bool atomic_block::binary_search_query(uint64_t item){

	int fill = get_fill();

	#if DEBUG_ASSERTS



	#if TAG_BITS == 8

	assert(short_byte_assert_sorted(tags, fill));


	#elif TAG_BITS == 16

	assert(sixteen_byte_assert_sorted(tags, fill));

	#endif


	#endif


	#if TAG_BITS == 8

	uint8_t tag = item & 0xff;

	#elif TAG_BITS == 16

	uint16_t tag = item & 0xffff;

	#endif



	int lower = 0;

	int upper = fill;

	int index;


	while (upper != lower){

		index = lower + (upper - lower)/2;


		int query_item = tags[index];

		if (query_item < tag){

			lower = index+1;

		} else if (query_item > tag){

			upper = index;

		} else {

			return true;
		}


	}

	if (lower < fill && tags[lower] == tag) return true;

	return false;





}

#endif //atomic_block_CU